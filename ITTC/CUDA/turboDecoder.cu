#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "main.h"
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
using namespace std;

#define M	3	// register length,=tail length
#define L_TOTAL 6144// if u want to use block interleave,L_TOTAL must = x^2
#define L_TOTAL_NUM 6147 
#define MAXITER 15
#define	FRAME_NUM 10000
//#define AlphaBetaBLOCK_NUM 8
//#define AlphaBetaTHREAD_NUM 8

//#define THREAD_NUM 8
#define BLOCK_NUM 12
#define L_BLOCK (L_TOTAL/BLOCK_NUM/4)
dim3 gridSize(2, BLOCK_NUM);
dim3 blockSize(4, 8);

#define LEAVER_BLOCK 8
#define LEAVER_THREAD 768
#define NSTATE	8	// = M^2
#define L_ALL (3*L_TOTAL+4*M)	// coded frame length

/*==================================================*/
/* ������� */
#define TERMINATED          1               /* 0����β��1��β */

#define TYPE_DECODER		1				/* ����������:1-LogMAP����
														  2-MAX-LogMAP����
														  3-SOVA����
                                                          4-const_LogMAP */
//#define N_ITERATION			8				/* ����������� */
#define MAX_FRAME_LENGTH	10000			/* ���֡�� */
/*==================================================*/
/* ��������� */
#define COLUMN_OF_G		4					/* ���������� */
#define G_ROW_1			13                  /* ������ͷ */					 
#define G_ROW_2			15					/* �����ͷ */

/*==================================================*/
/*==================================================*/

/* �������ݽṹ */

/* ������ṹ */
typedef struct
{
	int N_num_row;							/* ���������� */
	int K_num_col;							/* ���������� */
	int *g_matrix;							/* ��������ַ */
} TURBO_G;

/* Trellis�ṹ */
/*
mx_nextout(mx_lastout)Ϊ��(ǰ)�����������ַ:
����:״̬��, ����:4
ÿ�е�һ�к͵�����Ϊ��(ǰ)�������(1��-1),�ڶ��к͵�����Ϊ��֮��Ӧ�����(1��-1).

mx_nextstat(mx_laststat)Ϊ��(ǰ)��״̬������ַ:
����:״̬��, ����:2
���б�ʾ����Ϊ1(0)ʱ��Ӧ�ĺ�(ǰ)��״̬.
*/
typedef struct
{
	int *mx_nextout;		/* ����������� */						
	int *mx_nextstat;		/* ����״̬���� */
	int *mx_lastout;		/* ǰ��������� */
	int *mx_laststat;		/* ǰ��״̬���� */
	
} TURBO_TRELLIS;

/*==================================================*/
/* ����� */
#ifndef INFTY
#define INFTY 1E20
#endif
/*==================================================*/

/* ȫ�ֱ��� */

int *index_randomintlvr;		/* ҵ����Ϣ�����֯���±� */

int *index_randomintlvr2;

TURBO_G turbo_g;				/* ������ */

TURBO_TRELLIS turbo_trellis;	/* Tellis�ṹ */	

double rate_coding;				/* �������� */

void gen_qpp_index(int length, int *index);
/*==================================================*/
/* ����������صĲ��� */
int M_num_reg = COLUMN_OF_G-1;		/* �Ĵ����� */
int n_states = 8;						/* ״̬��:2��M_num_reg���� */
/*==================================================*/

int length_after_code;
int source_length;
int MODULATION;
int SYMBOL_NUM;
int f1,f2;
double _bpsk_map_i[2]=
{
	-1,1
};

double _bpsk_map_q[2]=
{
	0,0
};

__device__ float maxL(float x, float y) {
	return x>y?x:y;
}

__device__ float maxArray(float* arr, UINT length) {
	float temp = arr[0];
	for (int i = 1; i< length; i++) {
		if (arr[i] > temp)
			temp = arr[i];
	}
	return temp;
}
/*---------------------------------------------------------------
����:
	double E_algorithm(double x, double y)
����:
	Log-MAP�е�E�㷨:log(exp(x) + exp(y)) = max(x,y)+f(-|y-x|).
����:
	�������:
		x,y - ������.
	�������:
		��
����ֵ:
	�����
---------------------------------------------------------------*/
__device__ float E_algorithm(float x, float y)
{
	const double lookup_index_Log_MAP[16] = {0.0, 0.08824, 0.19587, 0.31026, 0.43275, 0.56508,
								0.70963, 0.86972, 1.0502, 1.2587, 1.5078, 1.8212,
								2.2522, 2.9706, 3.6764, 4.3758};
	const double lookup_table_Log_MAP[16] = {0.69315, 0.65, 0.6, 0.55, 0.5, 0.45, 0.4, 0.35,
								0.3, 0.25, 0.2, 0.15, 0.1, 0.05, 0.025, 0.0125};
	float temp = (y-x)>0? (y-x):(x-y);
	int i;

	if (temp>=4.3758)
	{
		temp = 0;
	}
	else
	{
		/* �����±� */
		for (i=0; i<16 && temp>=lookup_index_Log_MAP[i]; i++)
		{
			;
		}
		/* ����f(-|y-x|) */
		temp = (float)lookup_table_Log_MAP[i-1];
	}
	
	/* ����max(x,y)+f(-|y-x|) */
	return ( (x>y?x:y) + temp );
}	

/*---------------------------------------------------------------
����:
	double E_algorithm_seq(double *data_seq, int length)
����:
	���е�E�㷨.
����:
	�������:
		data_seq - ������ַ.
		length - ���г���
	�������:
		��
����ֵ:
	�����
---------------------------------------------------------------*/

__device__ float E_algorithm_seq(float *data_seq, int length)
{
	int i;			/* ѭ������ */
	float temp;

	/* ÿ��������E�㷨,������һ������E�㷨 */
	temp = E_algorithm(*(data_seq+0), *(data_seq+1));
	for (i=2; i<length; i++)
	{
		temp = E_algorithm(temp, *(data_seq+i));
	}
	return temp;
}
//////////////////////////////////////////////////////////////////////
// LogMAP component decoder
// index true decoder1 false decoder2
//////////////////////////////////////////////////////////////////////
__global__ void logmap(float *msg, float* parity, float* L_a, float* L_all)
{
    const char NextOut[2][NSTATE] = // check bit based on current and input bit
    {	-1,-1,1,1,1,1,-1,-1,
        1,1,-1,-1,-1,-1,1,1
    };
    // NextState[bk][current state]
    const BYTE NextState[2][NSTATE] = // next state based on current and input bit
    {	0,4,5,1,2,6,7,3,
        4,0,1,5,6,2,3,7
    };
    // LastOut[bk][current state]
    //const char LastOut[2][NSTATE] =	// trellis last check bit
    //{	-1,1,1,-1,-1,1,1,-1,
    //    1,-1,-1,1,1,-1,-1,1
    //};
    // LastState[bk][current state]
    const BYTE LastState[2][NSTATE] =	// last state lead to current state by input bk
    {	0,3,4,7,1,2,5,6,
        1,2,5,6,0,3,4,7
    };

    //const unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int block = 4*(blockIdx.x*BLOCK_NUM + blockIdx.y) + threadIdx.x;
	//const unsigned int decoderIndex = unsigned int(block/BLOCK_NUM);
    const unsigned int threadX = threadIdx.x;
	//const unsigned int blockNum = (unsigned int)(threadInBlock/8);
	const unsigned int threadY = threadIdx.y;
	const unsigned int kIndex = blockIdx.x*(6144+3) + (block%(BLOCK_NUM*4))*L_BLOCK;

	float gamma0, gamma1;

	INT k;

	__shared__ float Alpha[L_BLOCK+4][4][8];
	__shared__ float Beta[2][4][8];

	__shared__ float tempSum0[4][8];
	__shared__ float tempSum1[4][8];

	// initialize Alpha & Beta
	if ((block == 0 || block == BLOCK_NUM*4)&& threadY != 0) {
			Alpha[0][threadX][threadY]=-INFTY;
	}
	else {
			Alpha[0][threadX][threadY] = 0;
	}

	// forward recursion,compute Alpha 
	for (k=1;k<L_BLOCK+4;k++)
	{
        gamma0=-msg[kIndex + k-1]+parity[kIndex + k-1]*NextOut[0][LastState[0][threadY]]
            -L_a[kIndex + k-1]/2;
        gamma1=msg[kIndex + k-1]+parity[kIndex + k-1]*NextOut[1][LastState[1][threadY]]
            +L_a[kIndex + k-1]/2;

		Alpha[k][threadX][threadY] = 
			maxL(gamma0 + Alpha[k-1][threadX][LastState[0][threadY]], 
				gamma1 + Alpha[k-1][threadX][LastState[1][threadY]]);
	}

	// backward recursion,compute Beta
    if ((block == BLOCK_NUM*4 - 1 || block == BLOCK_NUM*8 -1) && threadY != 0){
        Beta[1][threadX][threadY] = -INFTY;
    }
    else
        Beta[1][threadX][threadY] = 0;

    if (block == BLOCK_NUM*4 - 1 || block == BLOCK_NUM*8 -1){
		for (k=L_BLOCK+2;k>L_BLOCK-1;k--){
			gamma0=-msg[kIndex + k]+parity[kIndex + k]*NextOut[0][threadY]
				-L_a[kIndex + k]/2;
			gamma1=msg[kIndex + k]+parity[kIndex + k]*NextOut[1][threadY]
				+L_a[kIndex + k]/2;

			Beta[0][threadX][threadY] = 
				maxL(gamma0 + Beta[1][threadX][NextState[0][threadY]], 
					gamma1 + Beta[1][threadX][NextState[1][threadY]]);

			__syncthreads();
			gamma0=-msg[kIndex + k]+parity[kIndex + k]*NextOut[0][LastState[0][threadY]]
				-L_a[kIndex + k]/2;
			gamma1=msg[kIndex + k]+parity[kIndex + k]*NextOut[1][LastState[1][threadY]]
				+L_a[kIndex + k]/2;

        	tempSum0[threadX][threadY] = gamma0+Alpha[k][threadX][LastState[0][threadY]]+Beta[1][threadX][threadY];

        	tempSum1[threadX][threadY] = gamma1+Alpha[k][threadX][LastState[1][threadY]]+Beta[1][threadX][threadY];

			Beta[1][threadX][threadY]=Beta[0][threadX][threadY];
        	__syncthreads();

			if (threadY == 0) {
				L_all[kIndex + k]= maxArray(*(tempSum1+threadX), 8) - maxArray(*(tempSum0+threadX), 8); 
			}
		}
    } 

	for (k=L_BLOCK-1;k>=0;k--)
	{
		gamma0=-msg[kIndex + k]+parity[kIndex + k]*NextOut[0][threadY]
			-L_a[kIndex + k]/2;
		gamma1=msg[kIndex + k]+parity[kIndex + k]*NextOut[1][threadY]
			+L_a[kIndex + k]/2;

		Beta[0][threadX][threadY] = 
			maxL(gamma0 + Beta[1][threadX][NextState[0][threadY]], 
				gamma1 + Beta[1][threadX][NextState[1][threadY]]);

		__syncthreads();

		gamma0=-msg[kIndex + k]+parity[kIndex + k]*NextOut[0][LastState[0][threadY]]
			-L_a[kIndex + k]/2;
		gamma1=msg[kIndex + k]+parity[kIndex + k]*NextOut[1][LastState[1][threadY]]
			+L_a[kIndex + k]/2;

        tempSum0[threadX][threadY] = gamma0+Alpha[k][threadX][LastState[0][threadY]]+Beta[1][threadX][threadY];
        tempSum1[threadX][threadY] = gamma1+Alpha[k][threadX][LastState[1][threadY]]+Beta[1][threadX][threadY];

		Beta[1][threadX][threadY]=Beta[0][threadX][threadY];
        __syncthreads();

        if (threadY == 0) {
            L_all[kIndex + k]= maxArray(*(tempSum1+threadX), 8) - maxArray(*(tempSum0+threadX), 8); 
        }
	}
}

//__global__ void interLeave(float * src, float * des){
//    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
//    des[tid] = src[(((263 + tid*480)%6144)*tid)%6144];
//    des[(((263 + tid*480)%6144)*tid)%6144 + 6144] = src[tid+6144];
//}

__global__ void deInterLeave(float * src, float * des){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    //des[interLeaveTable[tid]] = src[tid];
    des[(((263 + tid*480)%6144)*tid)%6144] = src[tid];
    des[tid + 6144+3] = src[(((263 + tid*480)%6144)*tid)%6144 + 6144 +3];

	if (tid == 0){
	for (int i = 0; i < M; i++){
	    des[6144+i] = 0;
	    des[6144+3+6144+i] = 0;
	    }
	}
}

__global__ void extrinsicInformation(float * L_all, float * msg, float * L_a, float * L_e) {
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    L_e[tid] = L_all[tid + 6144+3] - 2*msg[tid + 6144+3] - L_a[tid + 6144+3];
    L_e[tid + 6144+3] = L_all[tid] - 2*msg[tid] - L_a[tid];

    if (tid == 0){
		for (int i = 0; i < 3; i++){
			L_e[6144+i] = L_all[6144+3+6144+i] - 2*msg[6144+3+6144+i] - L_a[6144+3+6144+i];
			L_e[6144 + 6144+3+i] = L_all[6144+i] - 2*msg[6144+i] - L_a[6144+i];
		}
    }
}

__global__ void demultiplex(float * stream, float * msg, float * parity) {
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    //if (puncture){// punctured rate=1/2
    //    msg[tid]=stream[2*tid];
    //    parity[tid%2][tid]=stream[tid*2+1];
    //}
    //else {// unpunctured rate=1/3
    //    msg[tid]=stream[3*tid];
    //    parity0[tid]=stream[3*tid+1];
    //    parity1[tid]=stream[3*tid+2];
    //}
        msg[tid]=0.5*stream[3*tid];
		msg[6144 + 3 + tid] = 0.5*stream[3*((((263 + tid*480)%6144)*tid)%6144)];
        parity[tid]=0.5*stream[3*tid+1];
        parity[6144+3+tid]=0.5*stream[3*tid+2];

		if (tid == 0){
		    for (int i = 0; i<M; i++){
		        msg[6144+i] = 0.5*stream[3*6144+2*i];
		        parity[6144+i] = 0.5*stream[3*6144+2*i+1];
				msg[6144 + 3+6144 + i] = 0.5*stream[3*6144+2*3+2*i];
				parity[6144 + 3+6144 + i] = 0.5*stream[3*6144+2*3+2*i+1];
			}
		}
}

__global__ void initializeExtrinsicInformation(float * L_e) {
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    L_e[tid] = 0;
	L_e[6144+3+tid] = 0;

	if (tid == 0){
	    for (int i = 0; i < M; i++){
	        L_e[6144+i] = 0;
	        L_e[6144+3+6144+i] = 0;
		}
	}
}

__global__ void exestimateInformationBits(float * L_all, int * msghat) {
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(L_all[tid + 6144+3]>0)
        //msghat[m_Inter_table[tid]]=1;
		msghat[(((263 + tid*480)%6144)*tid)%6144] = 1;
    else
        //msghat[m_Inter_table[tid]]=0;
		msghat[(((263 + tid*480)%6144)*tid)%6144] = 0;
}

void countErrors(int *m, int * mhat, UINT * bitsError, UINT * frameError, UINT iter) {

	bool f_err = false;
	for (int i=0; i<(L_TOTAL);i++) {
		if (m[i] != mhat[i]) {
			bitsError[iter] = bitsError[iter]+1;
			f_err = true;
		}
	}

	if (f_err) 
		frameError[iter] = frameError[iter]+1;
}

int main(int argc, char* argv[])
{
	MODULATION = 1;			//���ƽ�����1,2,3,4,6
	source_length = 6144;			//������Դ����
	length_after_code = 6144*3+12;//SYMBOL_NUM*MODULATION;	//������׺�ĳ���	
	//SYMBOL_NUM=length_after_code/MODULATION;//  atoi(argv[4]);//672
	SYMBOL_NUM=length_after_code;

	f1 =263;
	f2 =480; 					//���볤�йصĽ�֯������

	double EbN0start = 0;//0;		//������ʼ�����
	double EbN0end = 1;//4;		//��������ֹ�����
	double EbN0step =0.1;		//��������Ȳ���
		
	double rate = (double)source_length/(double)(SYMBOL_NUM);

	int *source = NULL;
	int *mhat = NULL;
	clock_t start, finish;
	long double duration[FRAME_NUM];

	UINT bits_all,bits_err[MAXITER],frame_err[MAXITER];

	float Ber,Fer;

	int *coded_source = NULL;

	double *modulated_source_i, *modulated_source_q;

	double *after_channel_i,*after_channel_q;
	float *flow_for_decode = NULL;
	//int *flow_decoded = NULL;

	double EbN0dB,sigma;
	int nf, i1=0;

	TurboCodingInit();
			
	if ((source=(int *)malloc(source_length*sizeof(int)))==NULL)
	{
	  printf("\n fail to allocate memory of source \n");
	  exit(1);  
	}
	mhat=(int *)malloc(source_length*sizeof(int));

	if ((coded_source=(int *)malloc((3*source_length+4*M_num_reg)*sizeof(int)))==NULL)
	{
	  printf("\n fail to allocate memory of coded_source \n");
	  exit(1);  
	}

	if ((modulated_source_i=(double *)malloc((int)(SYMBOL_NUM)*sizeof(double)))==NULL)
	{
	  printf("\n fail to allocate memory of modulated_source_i \n");
	  exit(1);  
	}
	if ((modulated_source_q=(double *)malloc((int)(SYMBOL_NUM)*sizeof(double)))==NULL)
	{
	  printf("\n fail to allocate memory of modulated_source_q \n");
	  exit(1);  
	}
	if ((after_channel_i=(double *)malloc((int)(SYMBOL_NUM)*sizeof(double)))==NULL)
	{
	  printf("\n fail to allocate memory of after_channel_i \n");
	  exit(1);  
	}
	if ((after_channel_q=(double *)malloc((int)(SYMBOL_NUM)*sizeof(double)))==NULL)
	{
	  printf("\n fail to allocate memory of after_channel_q \n");
	  exit(1);  
	}

	if ((flow_for_decode=(float *)malloc((3*source_length+4*M_num_reg)*sizeof(float)))==NULL)
	{
	  printf("\n fail to allocate memory of flow_for_decode \n");
	  exit(1);  
	}
	//if ((flow_decoded=(int *)malloc(N_ITERATION*source_length*sizeof(int)))==NULL)
	//{
	//  printf("\n fail to allocate memory of flow_decoded\n");
	//  exit(1);  
	//}
	
	srand((unsigned)time(NULL));

	findCudaDevice(argc, (const char **)argv);

	float * yDevice;
	float * msgDevice;
	int * mhatDevice;
	float * parityDevice;
	float * L_eDevice;
	float * L_aDevice;
	float * L_allDevice;

    hipMalloc((void **)&yDevice, L_ALL*sizeof(float));
    hipMalloc((void **)&msgDevice, L_TOTAL_NUM*2*sizeof(float));
    hipMalloc((void **)&mhatDevice, L_TOTAL*sizeof(int));
    hipMalloc((void **)&parityDevice, L_TOTAL_NUM*2*sizeof(float));
    hipMalloc((void **)&L_eDevice, L_TOTAL_NUM*2*sizeof(float));
    hipMalloc((void **)&L_aDevice, L_TOTAL_NUM*2*sizeof(float));
    hipMalloc((void **)&L_allDevice, L_TOTAL_NUM*2*sizeof(float));

	for (EbN0dB=EbN0start; EbN0dB<=EbN0end; EbN0dB+=EbN0step)
	{
		sigma = pow(10,-EbN0dB/20)*sqrt(0.5/(rate*MODULATION));

		for (int i =0; i<MAXITER;i++) {
			bits_err[i]=0;
			frame_err[i]=0;
		}
		bits_all = 0;

		for (nf=0; nf<FRAME_NUM; nf++,bits_all += L_TOTAL)
		{
			for(i1=0; i1<source_length; i1+=1)
			{
				*(source+i1)=rand()%2;
			}

            TurboEnCoding(source, coded_source, source_length);

/*******************************************************************************/
			module(coded_source,modulated_source_i,modulated_source_q,SYMBOL_NUM*MODULATION,MODULATION);

			AWGN(modulated_source_i, after_channel_i, sigma, SYMBOL_NUM);
			AWGN(modulated_source_q, after_channel_q, sigma, SYMBOL_NUM);

			demodule(after_channel_i, after_channel_q, SYMBOL_NUM,flow_for_decode,1/(2*pow(sigma,2)),MODULATION);
/*******************************************************************************/

			hipMemcpy(yDevice,flow_for_decode,sizeof(float)*length_after_code, hipMemcpyHostToDevice);
			start = clock();

			demultiplex<<<LEAVER_BLOCK,LEAVER_THREAD>>>(yDevice, msgDevice, parityDevice); 
			initializeExtrinsicInformation<<<LEAVER_BLOCK,LEAVER_THREAD>>>(L_eDevice);

			for (int iter = 0; iter<MAXITER; iter++) {
				
				deInterLeave<<<LEAVER_BLOCK,LEAVER_THREAD>>>(L_eDevice, L_aDevice);

                logmap<<<gridSize, blockSize>>>(msgDevice, parityDevice, L_aDevice, L_allDevice);

				extrinsicInformation<<<LEAVER_BLOCK,LEAVER_THREAD>>>(L_allDevice, msgDevice, L_aDevice, L_eDevice);

				//interLeave<<<LEAVER_BLOCK,LEAVER_THREAD>>>(L_eDevice, L_aDevice);

                //logmap<<<BLOCK_NUM, THREAD_NUM>>>(imsgDevice, parity1Device, L_aDevice, L_allDevice, false);

				//extrinsicInformation<<<LEAVER_BLOCK,LEAVER_THREAD>>>(L_allDevice, imsgDevice, L_aDevice, L_eDevice);

				exestimateInformationBits<<<LEAVER_BLOCK,LEAVER_THREAD>>>(L_allDevice, mhatDevice); 

				hipMemcpy(mhat, mhatDevice, sizeof(int)*L_TOTAL, hipMemcpyDeviceToHost);
				countErrors(source, mhat, bits_err, frame_err, iter);

			}
			finish = clock();
			duration[nf] = (long double)(finish - start);

			//hipMemcpy(mhat, mhatDevice, sizeof(int)*L_TOTAL, hipMemcpyDeviceToHost);
			//countErrors(source, mhat, bits_err, frame_err, MAXITER-1);

		}
		printf("-------------------------\n");
		printf("Eb/No=%fdB:\n",EbN0dB);
		printf("-------------------------\n");

		for (int i=0;i<MAXITER;i++) {
			Ber=(float)bits_err[i]/(float)bits_all;
			Fer=(float)frame_err[i]/(float)FRAME_NUM;
			printf("Iteration:%d\n",i+1);
			printf("---Ber=%f\n---Fer=%f\n",Ber,Fer);
		}
		long double durationSum = 0.0;
		for (int i = 0; i < FRAME_NUM; i++) {
			durationSum += duration[i];
		}
		durationSum /= CLOCKS_PER_SEC;

		long double throughput = FRAME_NUM*(6144+3) / durationSum / 1000000;
		cout<<"throughput: "<<throughput<<"Mbps"<<endl;
			
	}
	
/*-----------------------------------------------------------------*/	
/*-----------------------------------------------------------------*/	
	TurboCodingRelease();
/*-----------------------------------------------------------------*/
/*-----------------------------------------------------------------*/	

	free(source);
	free(coded_source);
	free(modulated_source_i);
	free(modulated_source_q);
	free(after_channel_i);
	free(after_channel_q);
	free(flow_for_decode);
	//free(flow_decoded);

	hipFree(yDevice);
	hipFree(msgDevice);
	hipFree(mhatDevice);
	hipFree(parityDevice);
	hipFree(L_eDevice);
	hipFree(L_aDevice);
	hipFree(L_allDevice);
} //main


void randominterleaver_long(long *data_unintlvr, long *interleaverddata, int *index_randomintlvr, int length)
{
	int i;
	int *index_random = index_randomintlvr;

	for (i=0; i<length; i++)
	{
		*(interleaverddata+i) = *(data_unintlvr+ (*(index_random+i)));
	}
}

void random_deinterlvr_long(long *data_unintlvr, long *interleaverddata, int *index_randomintlvr, int length)
{
	int i;
	int *index_random = index_randomintlvr;

	for (i=0; i<length; i++)
	{
		*(data_unintlvr+(*(index_random+i))) = *(interleaverddata+i);
	}
}


void randominterleaver_int(int *data_unintlvr, int *interleaverddata, int *index_randomintlvr, int length)
{
	int i;
	int *index_random = index_randomintlvr;

	for (i=0; i<length; i++)
	{
		*(interleaverddata+i) = *(data_unintlvr+ (*(index_random+i)));
	}
}

void random_deinterlvr_int(int *data_unintlvr, int *interleaverddata, int *index_randomintlvr, int length)
{
	int i;
	int *index_random = index_randomintlvr;

	for (i=0; i<length; i++)
	{
		*(data_unintlvr+(*(index_random+i))) = *(interleaverddata+i);
	}
}

void randominterleaver_double(double *data_unintlvr, double *interleaverddata, int *index_randomintlvr, int length)
{
	int i;
	int *index_random = index_randomintlvr;

	for (i=0; i<length; i++)
	{
		*(interleaverddata+i) = *(data_unintlvr+ (*(index_random+i)));
	}
}

void random_deinterlvr_double(double *data_unintlvr, double *interleaverddata, int *index_randomintlvr, int length)
{
	int i;
	int *index_random = index_randomintlvr;

	for (i=0; i<length; i++)
	{
		*(data_unintlvr+(*(index_random+i))) = *(interleaverddata+i);
	}
}

/*---------------------------------------------------------------
����:
	int gen_g_matrix(int k_column, int g_row1, int g_row2, int *mx_g_turbo)
����:
	�õ�������.
����:
	�������:
		k_column - ����������.
		g_row1 - �������һ��.
		g_row2 - ������ڶ���.
	�������:
		mx_g_turbo - ��������ַ.
����ֵ:
	1 - �ɹ��صõ�������.
	0 - ����������ʧ��.
---------------------------------------------------------------*/
int gen_g_matrix(int k_column, int g_row1, int g_row2, int *mx_g_turbo)
{
	int i, position;		/* ѭ������ */
	int high_num, low_num;
	
	/* ��һ�� */
	high_num = g_row1;		
	position = 1;			/* �ڼ���8������ */
	while (high_num>0)
	{
		low_num = high_num%10;	/* �õ����8������ */
		if (low_num>7)			/* �ж��Ƿ�Ϊ8������ */
		{
			return 0;
		}
		high_num = high_num/10;		/* �������ಿ�� */

		/* ��8������תΪ�����Ʋ����� */
		for (i=k_column-(position-1)*3-1; i>=0 && i>=k_column-position*3; i--)
		{
			*(mx_g_turbo+i) = low_num%2;
			low_num = low_num/2;
		}
		position++;		/* ��һ��λ�� */
		if (i<0)
		{
			break;
		}
	}

	/* �ڶ��� */
	high_num = g_row2;
	position = 1;			/* �ڼ���8������ */
	while (high_num>0)
	{
		low_num = high_num%10;		/* �õ����8������ */
		if (low_num>7)				/* �ж��Ƿ�Ϊ8������ */
		{
			return 0;
		}
		high_num = high_num/10;		/* �������ಿ�� */

		/* ��8������תΪ�����Ʋ����� */
		for (i=k_column-(position-1)*3-1; i>=0 && i>=k_column-position*3; i--)
		{
			*(mx_g_turbo+k_column+i) = low_num%2;
			low_num = low_num/2;
		}
		position++;					/* ��һ��λ�� */
		if (i<0)
		{
			break;
		}
	}
	return 1;
}

/*---------------------------------------------------------------
����:
	void int2bin(int intstat, int *tempstat, int length)
����:
	ʮ������תΪ����������.
����:
	�������:
		intstat - ʮ������.
		length - Ҫ�õ��Ķ��������г��ȣ�
	�������:
		bin_stat - ������������ַ.
����ֵ:

	��
---------------------------------------------------------------*/
void int2bin(int intstat, int *bin_stat, int length)
{
	int i, temp;

	temp = intstat;

	/* ����2������ */
	for (i=length-1; i>=0; i--)
	{
		*(bin_stat+i) = temp%2;
		temp = temp/2;
	}
}

/*---------------------------------------------------------------
����:
	int bin2int(int *binseq, int length)
����:
	����������תΪʮ������.
����:
	�������:
		binseq - ������������ַ.
		length - ���������г��ȣ�
	�������:
		��
����ֵ:
	�õ���ʮ��������
---------------------------------------------------------------*/
int bin2int(int *binseq, int length)
{
	int i, j, temp;
	int sum = 0;

	for (i=0; i<length; i++)
	{
		temp = 1;

		/* �����λȨֵ */
		for (j=1; j<=i; j++)
		{
			temp = temp * 2;
		}
		sum = sum + temp * (*(binseq+length-1-i));
	}

	return sum;
}

/*---------------------------------------------------------------
����:
	int encode_bit(int inbit, int *stat)
����:
	���ر�����.
����:
	�������:
		inbit -�������.
		stat - ��ǰ�Ĵ���״̬��ַ.
	�������:
		stat - �����Ĵ���״̬��ַ.
����ֵ:
	������أ�
---------------------------------------------------------------*/
int encode_bit(int inbit, int *stat)
{
	int j;			/* ѭ������ */
	int output;		/* ������� */

	/* ����������� */
	output = (*(turbo_g.g_matrix+turbo_g.K_num_col+0)) * inbit;

	for (j=1; j<turbo_g.K_num_col; j++)
	{
		output = (output + (*(turbo_g.g_matrix+turbo_g.K_num_col+j)) * (*(stat+j-1)))%2;
	}

	/* �޸�״̬���� */
	for (j=turbo_g.K_num_col-2; j>0; j--)
	{
		*(stat+j)=*(stat+j-1);
	}

	*(stat+0) = inbit;

	return output;
}

/*---------------------------------------------------------------
����:
	void gen_trellis()
����:
	����Trellis.
����:
	��
����ֵ:
	��
---------------------------------------------------------------*/
void gen_trellis()
{
	int i, j, k;						/* ѭ������ */
	int dk_turbo, ak_turbo, outbit;		/* �������ڲ����غ�������� */

	int *tempstat;						/* ״̬���� */

	if ((tempstat=(int *)malloc(sizeof(int)*M_num_reg))==NULL)
	{
	  printf("\n fail to allocate memory of tempstat \n");
	  exit(1);  
	}
	/* ���ɺ�������ͺ���״̬���� */
	for (i=0; i<n_states; i++)			/* ״̬ѭ�� */
	{
		for (j=0; j<2; j++)				/* ����Ϊ0,1 */
		{
			int2bin(i, tempstat, M_num_reg);	/* ��״̬תΪ���������� */

			/* dk:ԭʼ������� */
			dk_turbo = j;

			/* ����ak:���ӷ������������أ��������ɾ����һ�У�13�� */
			ak_turbo = (*(turbo_g.g_matrix+0)) * dk_turbo;
			for (k=1; k<turbo_g.K_num_col; k++)
			{
				ak_turbo = ak_turbo + (*(turbo_g.g_matrix+k)) * (*(tempstat+k-1));
			}

			ak_turbo = ak_turbo % 2;

			/* �����������,�޸�״̬���У��������ɾ���ڶ��У�15�� */
			outbit = encode_bit(ak_turbo, tempstat);

			/* д���������ͺ���״̬���� */
			*(turbo_trellis.mx_nextout+i*4+2*j)=2*dk_turbo-1;
			*(turbo_trellis.mx_nextout+i*4+2*j+1)=2*outbit-1;
			*(turbo_trellis.mx_nextstat+i*2+j)=bin2int(tempstat, M_num_reg);
		}			/* ����ѭ������ */

	}	/* ״̬ѭ������ */

	/* ����ǰ�������ǰ��״̬���� */
	for (i=0; i<n_states; i++)	/* ״̬ѭ�� */
	{
		for (j=0; j<2; j++)/* ����Ϊ0,1 */
		{
			*(turbo_trellis.mx_laststat+(*(turbo_trellis.mx_nextstat+i*2+j))*2+j) = i;
			*(turbo_trellis.mx_lastout+(*(turbo_trellis.mx_nextstat+i*2+j))*4+2*j)
				= *(turbo_trellis.mx_nextout+i*4+2*j);
			*(turbo_trellis.mx_lastout+(*(turbo_trellis.mx_nextstat+i*2+j))*4+2*j+1)
				= *(turbo_trellis.mx_nextout+i*4+2*j+1);
		}	/* ����ѭ������ */
	}	/* ״̬ѭ������ */

	free(tempstat);
}

/*---------------------------------------------------------------
����:
	void TurboCodingInit()
����:
	Turbo���ʼ������.
����:
	��
����ֵ:
	��
---------------------------------------------------------------*/
void TurboCodingInit()
{
	/* ��ʼ�������� */
	turbo_g.N_num_row = 3;				/* ���� */
	turbo_g.K_num_col = COLUMN_OF_G;	/* ���� */

	SYMBOL_NUM = (int)((double)length_after_code/(MODULATION));

	if ((turbo_g.g_matrix=(int *)malloc(turbo_g.N_num_row*turbo_g.K_num_col*sizeof(int)))==NULL)
	{
		printf("\n fail to allocate memory of turbo_g\n");
		exit(1);  
	}

	/* �õ������� */
	if (!gen_g_matrix(turbo_g.K_num_col, G_ROW_1, G_ROW_2, turbo_g.g_matrix))
	{
		printf("error number of G\n");
		exit(1);
	}

	/* ����Trellis */
	if ((turbo_trellis.mx_lastout=(int *)malloc(sizeof(int)*n_states*4))==NULL)
	{
	  printf("\n fail to allocate memory of turbo_trellis.mx_lastout \n");
	  exit(1);  
	}
	if ((turbo_trellis.mx_laststat=(int *)malloc(sizeof(int)*n_states*2))==NULL)
	{
	  printf("\n fail to allocate memory of turbo_trellis.mx_laststat \n");
	  exit(1);  
	}
	if ((turbo_trellis.mx_nextout=(int *)malloc(sizeof(int)*n_states*4))==NULL)
	{
	  printf("\n fail to allocate memory of turbo_trellis.mx_nextout \n");
	  exit(1);  
	}
	if ((turbo_trellis.mx_nextstat=(int *)malloc(sizeof(int)*n_states*2))==NULL)
	{
	  printf("\n fail to allocate memory of turbo_trellis.mx_nextstat \n");
	  exit(1);  
	}

	gen_trellis();

	/* Ϊ�����֯�������ڴ� */
		if ((index_randomintlvr=(int *)malloc(MAX_FRAME_LENGTH*sizeof(int)))==NULL)
		{
			printf("\n fail to allocate memory of index_randomintlvr \n");
			exit(1);  
		}

		if ((index_randomintlvr2=(int *)malloc(2*MAX_FRAME_LENGTH*sizeof(int)))==NULL)
		{
			printf("\n fail to allocate memory of index_randomintlvr \n");
			exit(1);  
		}

	/*����QPP��֯���±�*/
	gen_qpp_index(source_length, index_randomintlvr);

	/*int i1,*num;

	if ((num=(int *)malloc(source_length*sizeof(int)))==NULL)
	{
		printf("\n fail to allocate memory of num \n");
		exit(1);  
	}

	for (i1=0;i1<source_length;i1++)
	{
		num[i1]=0;	
	}
	for (i1=0;i1<source_length;i1++)
	{
		num[index_randomintlvr[i1]]++;
			
	}

	for (i1=0;i1<source_length;i1++)
	{
	printf("%d ",num[i1]);
			
	}
    free(num);*/
}

/*---------------------------------------------------------------
����:
	void rsc_encode(int *source, int *rsc, int terminated, int len_info)
����:
	RSC������.
����:
	�������:
		source -Դ����������ַ.
		len_info - Դ�������г��ȣ�
		terminated - �Ƿ��β: 1-��β, 0-����β.
	�������:
		RSC - ���������������ַ.
����ֵ:
	�ޣ�
---------------------------------------------------------------*/
void rsc_encode(int *source, int *rsc, int terminated, int len_info)
{
	int i, j;			/* ѭ������ */

	int *state;			/* ״̬���� */
	int dk_turbo, ak_turbo, outbit;		/* �������ڵ�dk,ak�ͱ���������� */

	int len_total;						/* �ܳ��� */

	if ((state=(int *)malloc(M_num_reg*sizeof(int)))==NULL)
	{
	  printf("\n fail to allocate memory of state \n");
	  exit(1);  
	}

	/* �����ܳ��� */
	len_total = len_info+M_num_reg;

	/* ��״̬Ϊ0 */
	for (i=0; i<M_num_reg; i++)
	{
		*(state+i) = 0;
	}

	for (i=0; i<len_total; i++)							/* ����ر��� */
	{
		if (!terminated || (terminated && i<len_info))	/* ����Ϣ���� */
		{
			dk_turbo = *(source+i);
		}
		else											/* ��βTrellis */
		{
			if (terminated && i>=len_info)
			{
				dk_turbo = 0;
				for (j=1; j<turbo_g.K_num_col; j++)
				{
					dk_turbo = dk_turbo + (*(turbo_g.g_matrix+j)) * (*(state+j-1));
				}
				dk_turbo = dk_turbo%2;
			}
		}

		/* ����ak */
		ak_turbo = *(turbo_g.g_matrix+0) * dk_turbo;
		for (j=1; j<turbo_g.K_num_col; j++)
		{
			ak_turbo = ak_turbo + (*(turbo_g.g_matrix+j))*(*(state+j-1));
		}

		ak_turbo = ak_turbo%2;

		/* ��ak���б��ر��� */
		outbit = encode_bit(ak_turbo, state);

		/* дdk��������� */
		*(rsc+2*i) = dk_turbo;
		*(rsc+2*i+1) = outbit;
	}				/* ����ر������ */

	free(state);
}

/*---------------------------------------------------------------
����:
	void encoderm_turbo(int *source, int *send_turbo, int len_info)

����:
	Turbo�������.
����:
	�������:
		source -Դ����������ַ.
		len_info - Դ�������г��ȣ�
		type_flow - ��Ϣ����: 1-ҵ����Ϣ, 0������Ϣ.
	�������:
		send_turbo - ������ƺ�����������ַ.
����ֵ:
	�ޣ�
---------------------------------------------------------------*/
void encoderm_turbo(int *source, int *send_turbo, int len_info)
{
	int i;									/* ѭ������ */
	int len_total = len_info + M_num_reg;	/* �ܳ��� */

	int *rsc1, *rsc2;		/* ����RSC����������� */
	
	int *input2;			/* RSC2������ */

	if ((rsc1=(int *)malloc(2*len_total*sizeof(int)))==NULL)
	{
	  printf("\n fail to allocate memory of rsc1 \n");
	  exit(1);  
	}
	if ((rsc2=(int *)malloc(2*len_total*sizeof(int)))==NULL)
	{
	  printf("\n fail to allocate memory of rsc2 \n");
	  exit(1);  
	}

	if ((input2=(int *)malloc(len_info*sizeof(int)))==NULL)
	{
	  printf("\n fail to allocate memory of input2 \n");
	  exit(1);  
	}

	/* RSC1 */
	rsc_encode(source, rsc1, TERMINATED, len_info);

	/* ��֯��Դ���� */
	randominterleaver_int(source, input2, index_randomintlvr, len_info);

	/* RSC2 */
	rsc_encode(input2, rsc2, TERMINATED, len_info);

	/* ��Ϣλ���õ��� */
	for (i=0; i<len_info; i++)
	{
		*(send_turbo+3*i) = *(rsc1+2*i);
		*(send_turbo+3*i+1) = *(rsc1+2*i+1);
		*(send_turbo+3*i+2) = *(rsc2+2*i+1);
	}
	
	/* ��βλ���õ��� */
	for (i=0; i<2*M_num_reg; i++)
	{
		*(send_turbo+3*len_info+i) = *(rsc1+2*len_info+i);
		*(send_turbo+3*len_info+2*M_num_reg+i) = *(rsc2+2*len_info+i);
	}
	
	free(rsc1);
	free(rsc2);
	free(input2);
}


/*---------------------------------------------------------------
����:
	double random_turbo()
����:
	����0-1���ȷֲ������.
����:
	��
����ֵ:
	���ɵ�0-1���ȷֲ��������
---------------------------------------------------------------*/
double random_turbo()
{
	long z,k;
	static long s1 = 12345L;
	static long s2 = 1234546346L;

	k= s1 / 53668L;
	s1 = 40014L * (s1 - k*53668L) - k*12211L;
	if (s1<0)
	  s1 = s1 + 2147483563L;
	k = s2 / 52774;
	s2 = 40692L * (s2 - k*52774L) - k*3791L;
	if (s2<0)
        s2 = s2 + 2147483399L;
 	z=s1 - s2;
	if (z<1)
  	  z = z + 2147483562L;
	return (double) z / (double) 2147483563.0;
}

void gen_qpp_index(int length, int *index)
{
	int i;

	for (i=0; i<source_length; i++)
	{
		*(index+i) = (f1*i+(((f2*i)%source_length)*i)%source_length)%source_length; // ������ת����Ϊ�˷�ֹ��ֵ̫��������
	}
}

/*---------------------------------------------------------------
����:
	void gen_rand_index(int length, int type_flow)
����:
	���������֯�����±�.
����:
	�������:
		length - ��֯������.
		type_flow - ��Ϣ����: 1-ҵ����Ϣ, 0-������Ϣ.
	�������:
		��
����ֵ:
	��
---------------------------------------------------------------*/
void gen_rand_index(int length, int *index)
{
	int *index_random;			/* �������ɵ�0-1���ȷֲ���������� */
	double *tempindex;			/* ����ѡ�������֯��ַ */
	double tempmax;				/* ���ֵ */
	int selectedscr;			/* ѡ�е��±� */
	int i, j;					/* ѭ������ */

	if ((tempindex=(double *)malloc((length)*sizeof(double)))==NULL)
	{
	  printf("\n fail to allocate memory of tempindex \n");
	  exit(1);  
	}

	/* ����������ѡ�����֯�� */
	index_random = index;

	/* ���ɵ�0-1���ȷֲ���������� */
	for (i=0; i<length; i++)
	{
		*(tempindex+i) = random_turbo();
	}

	for (i=0; i<length; i++)	
	{
		/* �ҵ�tempindex�е����ֵ��Ӧ���±� */
		tempmax = 0.0;

		for (j=0; j<length; j++)
		{
			if (*(tempindex+j) >= tempmax)
			{
				tempmax = *(tempindex+j);
				selectedscr = j;
			}
		}

		/* ��֯����λ��Ϊ���±� */
		*(index_random+i) = selectedscr;

		/* tempindex��λ��0 */
		*(tempindex+selectedscr) = 0.0;
	}

	free(tempindex);
}

/*---------------------------------------------------------------
����:
	TurboEnCoding(int *source, int *coded_source,
						int *source_length)
����:
	Turbo��ҵ����Ϣ���뺯��.
����:
	�������: source - Դbit������ַ.
			  source_length - Դbit���г���.
	�������: coded_source - ������������ַ.
����ֵ:
	��
---------------------------------------------------------------*/
void TurboEnCoding(int *source, int *coded_source, int source_length)
{
	int i;							/* ѭ������ */

	int *temp_send = NULL;			
	int *send = NULL;				/* ����������ַ */

	int length_info = source_length;		/* ��Ϣλ���� */

	/* �����ڴ� */
	if ((send=(int *)malloc((3*length_info+4*M_num_reg)*sizeof(int)))==NULL)
	{
		printf("\n fail to allocate memory of send \n");
		exit(1);
	}

	/* ���������֯���±� */
//	gen_rand_index(length_info, index_randomintlvr);

	encoderm_turbo(source, send, length_info);	/* ���� */

	temp_send = send;

	/* д������� */
	for (i=0; i<(3*length_info+4*M_num_reg); i++)
	{
		*(coded_source+i) = *(temp_send+i);
	}

	free(send);
}

/*---------------------------------------------------------------
����:
	double get_max(double *data_seq, int length)
����:
	�õ����е����ֵ.
����:
	�������:
		data_seq - ������ַ.
		length - ���г���.
	�������:
		��
����ֵ:
	�����е����ֵ��
---------------------------------------------------------------*/
double get_max(double *data_seq, int length)
{
	int i;		/* ѭ������ */
	double temp;
	temp = *(data_seq+0);
	for (i=1; i<length; i++)
	{
		if (temp < *(data_seq+i))
		{
			temp = *(data_seq+i);
		}
	}

	return temp;
}







/*---------------------------------------------------------------
����:
	void decision(double *LLR_seq, int length, int *output)
����:
	�о�.
����:
	�������:
		LLR_seq - LLR������ַ.
		length - ���г���
	�������:
		output - �о������������ַ
����ֵ:
	�ޣ�
---------------------------------------------------------------*/
void decision(double *LLR_seq, int length, int *output)
{
	int i;

	for (i=0; i<length; i++)
	{
		/* С��0��0 */
		if (*(LLR_seq+i) < 0)
		{
			*(output+i) = 0;
		}
		/* ����0��1 */
		else
		{
			*(output+i) = 1;
		}
	}		
}


void dectobin(double *flow_for_change, int *flow_changed, int flow_len, int integer_len, int decimal_len)
{
    int i;/* ѭ������ */
	int total_len = integer_len + decimal_len;

		/* �����ڴ� */
	/*if ((flow_changed=(int *)malloc((flow_len)*sizeof(int)))==NULL)
	{
		printf("\n fail to allocate memory of receive_punc \n");
		exit(1);  
	}*/

	for (i=0; i<flow_len; i++)
	{
		*(flow_for_change+i) *= pow(2,decimal_len);
		*(flow_changed+i) = (int)(*(flow_for_change+i)+0.5);
		if (*(flow_changed+i)<-pow(2,total_len))
		{
			*(flow_changed+i)=(long)-pow(2,total_len);
		}
		else if (*(flow_changed)>(pow(2,total_len)-1))
		{
			*(flow_changed+i)=(long)pow(2,total_len+i)-1;
		}            //-__logf(1+__expf(L_a[block*L_BLOCK + k-1]));

		/*if (*(flow_changed+i)>=0)
		{
			*(flow_changed+i) = *(flow_changed+i);
		}
		else 
		{
			*(flow_changed+i) = *(flow_changed+i)+pow(2,total_len);
		}*/

	}
	
	/*ofstream flow_changed_f("flow_changed.txt");
	flow_changed_f<< "-----flow_changed�ļ���ʼ-----" << endl;
	for (i=0; i<flow_len; i++) flow_changed_f << flow_changed[i] << endl;

	flow_changed_f<< "-----flow_changed�ļ�����-----" << endl;*/
}


void TurboCodingRelease()
{
	/* �ͷ��������ڴ� */
	free(turbo_g.g_matrix);
	/* �ͷ�trellis�ṹ�ڴ� */
	free(turbo_trellis.mx_lastout);
	free(turbo_trellis.mx_laststat);
	free(turbo_trellis.mx_nextout);
	free(turbo_trellis.mx_nextstat);
	/* �ͷ������֯���ڴ� */
	free(index_randomintlvr);
	free(index_randomintlvr2);

}
/*---------------------------------------------------------------
����:
	void mgrns(double mean,double sigma,double seed,int n,double *a)
����:
	��������Ϊn�ĸ�˹�������.
����:
	�������:	mean - ��ֵ
				sigma - ��׼��
				seed - һ���������
	�������:	a - ����Ϊn�ĸ�˹�������.
����ֵ:
	��
---------------------------------------------------------------*/
void mgrns(double mean,double sigma,double seed,int n,double *a)
{ int i,k,m;
    double s,w,v,t;
    s=65536.0; w=2053.0; v=13849.0;
    for (k=0; k<=n-1; k++)
	{
		t=0.0;
		for (i=1; i<=12; i++)
        { 
			seed=seed*w+v; m=(int)(seed/s);
            seed=seed-m*s; t=t+(seed)/s;
        }/*�������ļ��޶���������Ӹ�˹�ֲ��������*/
        *(a+k)=mean+(double)(sigma*(t-6.0));
    }
    return;
}
/*---------------------------------------------------------------
����:
	void AWGN(int *send, double *r, double sigma, int totallength)
����:
	AWGN�ŵ�.
����:
	�������: send - int�ͷ���������ַ.
			  sigma - ������׼��.
			  totallength - ���г���.
	�������: r - ��AWGN�ŵ�����������е���ַ.
����ֵ:
	��
---------------------------------------------------------------*/
void AWGN(double *send, double *r, double sigma, int totallength)
{
	int i;
	double *noise = (double *)malloc(sizeof(double)*totallength);
	double seed =  (double)(3.0 - (double)((rand() & RAND_MAX)/(double)RAND_MAX)/10e6);
	mgrns(0,sigma,seed,totallength,noise);
	for(i=0; i<totallength; i++)
	{
		*(r+i) = (double)( *(send+i) + *(noise+i) );
		//*(r+i) = (double)( *(send+i) + 0 );
	}
	free(noise);
}

/************************************************************************/

double  calculate_sqr_dis(double pt1_r,double pt1_i,double pt2_r,double pt2_i)
{
	double dis_r,dis_i,sqr_dis;

	dis_r=pt1_r-pt2_r;
	
	dis_i=pt1_i-pt2_i;

	sqr_dis=dis_r*dis_r+dis_i*dis_i;

	return sqr_dis;


}

void _bpsk_module(int * a,double * outi,double * outq,int N)
{
	int i;

	for (i=0;i<N;i++)
	{
		*(outq+i)=0.0;
		
		if (*(a+i)==1) 
			*(outi+i)=1.0;
		else 
			*(outi+i)=-1.0;
	}

}


void module(int * a,double * outi,double * outq,int N, int modu_index)
{
	switch (modu_index)
	{
	    case 1: _bpsk_module(a,outi,outq,N);break;
	}

}


void _bpsk_demodule(double *symbol_i, double *symbol_q, int symbol_len,
					float *out,double Kf)
{
	int i,j;
	double min_sqr_dis1,min_sqr_dis2,sqr_dis1,sqr_dis2;

	for(i=0;i<symbol_len;i++)
	{
	/*the minimum square distance has nothing with the orders of calculation of sqr_dis1 and sqr_dis2*/

		min_sqr_dis1=0x7fffffffffff,min_sqr_dis2=0x7fffffffffff;

        for(j=0;j<2;j++)
		{
			if(j&1)
			{
			/*calculate Min{dj} where j in Sj*/
			sqr_dis1=calculate_sqr_dis(symbol_i[i],symbol_q[i],(double)_bpsk_map_i[j],(double)_bpsk_map_q[j]);
	
			if(sqr_dis1<min_sqr_dis1)
				min_sqr_dis1=sqr_dis1;
			}

			else
			{
				/*calculate Min{dj} where j in complement of Sj*/	
			sqr_dis2=calculate_sqr_dis(symbol_i[i],symbol_q[i],(double)_bpsk_map_i[j],(double)_bpsk_map_q[j]);

			if(sqr_dis2<min_sqr_dis2)
				min_sqr_dis2=sqr_dis2;

			}
		}
        out[i]= -Kf*(min_sqr_dis1-min_sqr_dis2);
	}
}

void demodule(double *symbol_i, double *symbol_q, int symbol_len,float* out,double Kf,int modu_index)
{
    switch(modu_index)
	{
	    case 1: _bpsk_demodule(symbol_i, symbol_q,symbol_len,out,Kf);break;
	}
}


