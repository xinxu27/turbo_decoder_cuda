#include "hip/hip_runtime.h"
// TurboDecoder : Defines the entry point for the console application.
#include "hip/hip_runtime_api.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <limits.h>
#include <iostream>
using namespace std;


#define IM1 2147483563
#define IM2 2147483399
#define AM (1.0/IM1)
#define IMM1 (IM1-1)
#define IA1 40014
#define IA2 40692
#define IQ1 53668
#define IQ2 52774
#define IR1 12211
#define IR2 3791
#define NTAB 32
#define NDIV (1+IMM1/NTAB)
#define EPS 1.7E-308 //MINDOUBLE
#define RNMX (1.0-EPS)
#define INIFINITY  1E+10

#define MIN 1E-300
#define L_TOTAL 6144 // if u want to use block interleave,L_TOTAL must = x^2
#define MAXITER 10
#define	FRAME_NUM 10
#define AlphaBetaTHREAD_NUM 4

#define THREAD_NUM 768
#define BLOCK_NUM 16

typedef unsigned char BYTE;
typedef int INT;
typedef unsigned int UINT;
typedef int BOOL;

UINT m_Inter_table[L_TOTAL];
UINT LTE_Inter_table[L_TOTAL][3] =
	{40,3,10,	48,7,12,    56,19,42,	64,7,16,    72,7,18,    80,11,20,   
    88,5,22,    96,11,24,   104,7,26,   112,41,84,  120,103,90, 128,15,32,  
    136,9,34,   144,17,108, 152,9,38,	160,21,120, 168,101,84, 176,21,44,  
    184,57,46,  192,23,48,  200,13,50,  208,27,52,  216,11,36,  224,27,56,  
    232,85,58,  240,29,60,  248,33,62,  256,15,32,  264,17,198, 272,33,68,  
    280,103,210,288,19,36,  296,19,74,  304,37,76,  312,19,78,  320,21,120, 
    328,21,82,  336,115,84, 344,193,86, 352,21,44,  360,133,90, 368,81,46,  
    376,45,94,  384,23,48,  392,243,98, 400,151,40, 408,155,102,416,25,52,  
    424,51,106, 432,47,72,  440,91,110, 448,29,168, 456,29,114, 464,247,58, 
    472,29,118, 480,89,180, 488,91,122, 496,157,62, 504,55,84,  512,31,64,  
    528,17,66,  544,35,68,  560,227,420,576,65,96,  592,19,74,  608,37,76,  
    624,41,234, 640,39,80,  656,185,82, 672,43,252, 688,21,86,  704,155,44, 
    720,79,120, 736,139,92, 752,23,94,  768,217,48, 784,25,98,  800,17,80,  
    816,127,102,832,25,52,  848,239,106,864,17,48,  880,137,110,896,215,112,
    912,29,114, 928,15,58,  944,147,118,960,29,60,  976,59,122, 992,65,124, 
    1008,55,84, 1024,31,64, 1056,17,66, 1088,171,204,1120,67,140,1152,35,72,
    1184,19,74, 1216,39,76, 1248,19,78, 1280,199,240,1312,21,82,1344,211,252,
    1376,21,86, 1408,43,88, 1440,149,60,1472,45,92, 1504,49,846,1536,71,48, 
    1568,13,28, 1600,17,80, 1632,25,102,1664,183,104,1696,55,954,1728,127,96,
    1760,27,110,1792,29,112,1824,29,114,1856,57,116,1888,45,354,1920,31,120,
    1952,59,610,1984,185,124,2016,113,420,2048,31,64,2112,17,66,2176,171,136,
    2240,209,420,2304,253,216,2368,367,444,2432,265,456,2496,181,468,2560,39,80,
    2624,27,164,2688,127,504,2752,143,172,2816,43,88,2880,29,300,2944,45,92,
    3008,157,188,3072,47,96,3136,13,28,3200,111,240,3264,443,204,3328,51,104,
    3392,51,212,3456,451,192,3520,257,220,3584,57,336,3648,313,228,3712,271,232,
    3776,179,236,3840,331,120,3904,363,244,3968,375,248,4032,127,168,4096,31,64,
    4160,33,130,4224,43,264,4288,33,134,4352,477,408,4416,35,138,4480,233,280,
    4544,357,142,4608,337,480,4672,37,146,4736,71,444,4800,71,120,4864,37,152,
    4928,39,462,4992,127,234,5056,39,158,5120,39,80,5184,31,96,5248,113,902,
    5312,41,166,5376,251,336,5440,43,170,5504,21,86,5568,43,174,5632,45,176,
    5696,45,178,5760,161,120,5824,89,182,5888,323,184,5952,47,186,6016,23,94,
    6080,47,190,6144,263,480};

//typedef enum __bool { false = 0, true = 1, } bool;

long idum2;
long idum;
long iy;
long iv[NTAB];	
unsigned memory;

/*
Long period (? 2 \Theta 10 18 ) random number generator of L'Ecuyer with Bays­Durham shuffle
and added safeguards. Returns a uniform random deviate between 0.0 and 1.0 (exclusive of
the endpoint values). Call with idum a negative integer to initialize; thereafter, do not alter
idum between successive deviates in a sequence. RNMX should approximate the largest floating
value that is less than 1.
--
*/


double ran2()
{
	int j;
	long k;
	double temp;
	
	
	k=(idum)/IQ1;
	idum=IA1*(idum-k*IQ1)-k*IR1;  // Compute idum=(IA1*idum) % IM1 without overflows by Schrage's method.
	if (idum < 0)
		idum += IM1;
	k=idum2/IQ2;
	idum2=IA2*(idum2-k*IQ2)-k*IR2;	// Compute idum2=(IA2*idum) % IM2 likewise.
	if (idum2 < 0)
		idum2 += IM2;
	j = iy/NDIV;
	iy=iv[j]-idum2;
	//	iy=iv[j]-idum2; 		// Here idum is shuffled, idum and idum2 are combined to generate output.
	iv[j] = idum;
	if (iy < 1)
		iy += IMM1;
	if ((temp=AM*iy) > RNMX)
		return RNMX; 		// Because users don't expect endpoint values.
	else
		return temp;
}


void initRandom(long seed)
{
	idum2=123456789;
	idum=0;
	iy=0;
	
	if (seed != 0)
		idum = seed;
	else
		idum = 1;
	
	
    int j;
	for (j=NTAB+7;j>=0;j--) // Load the shuffle table (after 8 warm­ups).
	{
		long k=(idum)/IQ1;
		
		idum=IA1*(idum-k*IQ1)-k*IR1;
		if (idum < 0)
			idum += IM1;
		if (j < NTAB)
			iv[j] = idum;
	}
	iy=iv[0];
}


double doublerandom()
{
	double t = ran2();
	return t;
}

long longrandom(long range)
{
	double t;
	
	t = doublerandom();
	return((long)(t*(double)range));
}


bool boolrandom()
{
	double t=doublerandom();
	
	if (t>0.5)
		return true;
	else
		return false;
}
//time_t t;	
//time(&t);	
//init((long)t);
long seed = 1234421;





#define M	3	// register length,=tail length
#define NSTATE	8	// = M^2
#define L_ALL 3*L_TOTAL	// coded frame length
//#define DELTA 30	// SOVA window size. Make decision after 'delta' delay. Decide bit k when received bits
					// for bit (k+delta) are processed. Trace back from (k+delta) to k. 
// Rate 1/3 Turbo code
// The generator polynomials used are:
//	g0=[1 1 1]
//  g1=[1 0 1]
//	RSC encoder structure:
//
//			 +------------------------------------------>c1
//           |          g0(x)    
//           x_.--o-------------(+)<----------+
//           |    |             /|\           |
//			 |   \|/      ---    |     ---    |
// data --_.-o-->(+)--o->| D0|---o--->| D1|---o
//                    |   ---          ---    |
//                    |                       |
//                    +-------->(+)<----------+
//                      g1(x)    |
//								 +---------------------->c2
//

// NextOut[bk][current state]
static const char EnNextOut[2][NSTATE] = // check bit based on current and input bit
{	0,0,1,1,1,1,0,0,
	0,0,1,1,1,1,0,0
};
static const char NextOut[2][NSTATE] = // check bit based on current and input bit
{	-1,-1,1,1,1,1,-1,-1,
	-1,-1,1,1,1,1,-1,-1
};
// NextState[bk][current state]
static const BYTE NextState[2][NSTATE] = // next state based on current and input bit
{	0,4,5,1,2,6,7,3,
	4,0,1,5,6,2,3,7
};
// LastOut[bk][current state]
static const char LastOut[2][NSTATE] =	// trellis last check bit
{	-1,1,1,-1,-1,1,1,-1,
	-1,1,1,-1,-1,1,1,-1
};
// LastState[bk][current state]
static const BYTE LastState[2][NSTATE] =	// last state lead to current state by input bk
{	0,3,4,7,1,2,5,6,
	1,2,5,6,0,3,4,7
};
// TailBit[current state]
static const char TailBit[NSTATE] = // tail info bits when trellis is terminating
{	0,1,1,0,0,1,1,0
};








double gaussian(double variance)
{
	// static becuase we don't want to have it initialized each time we go in
	double returnvalue=0;
	double k;
	
	k = sqrt(variance/2.0);
	
	// add 24 uniform RV to obtain a simulation of normality
    int x;
	for (x=0;x<24;x++)
		returnvalue += doublerandom();
	
	return k*(returnvalue-0.5*24);

}




//////////////////////////////////////////////////////////////////////
// block interleave
// L_TOTAL must = x^2,otherwise,who knows?
//////////////////////////////////////////////////////////////////////
void init_Block_interleave_table()
{
	INT i,j;
	INT temp;

	temp = (INT)sqrt(L_TOTAL);
	for (i=0;i<temp;i++)
		for (j=0;j<temp;j++)
			m_Inter_table[i*temp+j] = j*temp+i;

	
}
void init_my_interleave_table() {
	INT i;
	for (i=0;i<L_TOTAL;i++)
		m_Inter_table[i] = L_TOTAL-1-i;
}

//////////////////////////////////////////////////////////////////////
// RSC endcoder
// mesg -- {0,1}
// parity -- {0,1}
// force==1,terminated --- for outer encoder
//////////////////////////////////////////////////////////////////////
void RSC_Encode(BYTE *mesg, BYTE *parity, unsigned int size, bool force)
{
	BYTE state,uk;
	unsigned x;
	
	state=0;
	for (x=0;x<size;x++)
	{
		// force the encoder to zero state at the end
		if (x>=size-M && force)
		{
			mesg[x] = TailBit[state];
		}
		
		// can't assume the bool type has an intrinsic value of 0 or 1
		// may differ from platform to platform
		uk = mesg[x] ? 1 : 0;
		
		// calculate output due to new mesg bit
		parity[x] = EnNextOut[uk][state];
		// calculate the new state
		state = NextState[uk][state];
	}
}


//////////////////////////////////////////////////////////////////////
// Turbo encoder
// msg -- {0,1}
// stream -- {0,1}
// puncture -- true to get 1/2 rate,NOT tested yet
//////////////////////////////////////////////////////////////////////
void encode(BYTE *msg, BYTE *stream, bool puncture)
{
	INT i;
	BYTE imsg[L_TOTAL];
	BYTE chkBuffer[2][L_TOTAL];
	// first encoder
	RSC_Encode(msg,chkBuffer[0],L_TOTAL,true);
	// interleave
	for (i=0;i<L_TOTAL;i++)
		imsg[i]=msg[m_Inter_table[i]];
	// second encoder
	RSC_Encode(imsg,chkBuffer[1],L_TOTAL,false);
	// punture
	for (i=0;i<L_TOTAL;i++)
	{
		if(puncture){
			stream[i*2]=msg[i];
			stream[i*2+1]=chkBuffer[i%2][i];
		}else{
			stream[i*3]=msg[i];
			stream[i*3+1]=chkBuffer[0][i];
			stream[i*3+2]=chkBuffer[1][i];
		}	
	}
}


__global__ void interLeave(double * src, double * des , unsigned int * interLeaveTable[3] ){
//__global__ void interLeave(double * src, double * des , unsigned int * interLeaveTable ){
    const int tid = threadIdx.x;
   //des[tid] = src[interLeaveTable[tid]];
    des[tid] = src[(interLeaveTable[tid][1]*tid+interLeaveTable[tid][2]*tid*tid)%interLeaveTable[tid][0]];
}

__global__ void deInterLeave(double * src, double * des , unsigned int * interLeaveTable[3] ){
//__global__ void deInterLeave(double * src, double * des , unsigned int * interLeaveTable ){
    const int tid = threadIdx.x;
    //des[interLeaveTable[tid]] = src[tid];
    des[(interLeaveTable[tid][1]*tid+interLeaveTable[tid][2]*tid*tid)%interLeaveTable[tid][0]] = src[tid];
}

__global__ void gammaAlpha(double * msg ,double * parity, double * L_a, double (*gamma)[8][8], BYTE (*lastState)[8],char (*lastOut)[8] ){
    const int tid = threadIdx.x;

    unsigned int s0,s2;
    for (s0=0;s0<NSTATE;s0++) {
		for (s2=0;s2<NSTATE;s2++)
			gamma[tid][s0][s2]=-INIFINITY;
		gamma[tid][s0][lastState[0][s0]]=-msg[tid]+parity[tid]*lastOut[0][s0]-log(1+exp(L_a[tid]));
		gamma[tid][s0][lastState[1][s0]]=msg[tid]+parity[tid]*lastOut[1][s0]+L_a[tid]-log(1+exp(L_a[tid]));
		//gamma[tid][s0][lastState[0][s0]]=0.5;
		//gamma[tid][s0][lastState[1][s0]]=-0.5;
    }
}

__global__ void gammaBeta(double * msg ,double * parity, double * L_a, double (*gamma)[8][8], BYTE (*nextState)[8], char (*nextOut)[8]){
    const int tid = threadIdx.x;

    unsigned int s0,s2;
    for (s0=0;s0<NSTATE;s0++) {
		for (s2=0;s2<NSTATE;s2++)
			gamma[tid][s0][s2]=-INIFINITY;
		gamma[tid][s0][nextState[0][s0]]=-msg[tid]+parity[tid]*nextOut[0][s0]-log(1+exp(L_a[tid]));
		gamma[tid][s0][nextState[1][s0]]=msg[tid]+parity[tid]*nextOut[1][s0]+L_a[tid]-log(1+exp(L_a[tid]));
		//gamma[tid][s0][nextState[0][s0]]=0.5;
		//gamma[tid][s0][nextState[1][s0]]=-0.5;
    }
}

__global__ void Alpha(double (*Alpha)[8], double (*gamma)[8][8], double *maxBranch) {
	const int tid = threadIdx.x;

	UINT k, s1, s2;
	double sum;

	if (tid == 0) {
		Alpha[0][0] = 0.0;
		for (s1=1;s1<NSTATE;s1++)
			Alpha[0][s1]=-INIFINITY;
	}
	else {
		for (s1=0;s1<NSTATE;s1++)
			Alpha[tid*(L_TOTAL/AlphaBetaTHREAD_NUM)][s1]=0;
	}

	//for (k=1; k<=L_TOTAL; k++) {
	for (k=tid*L_TOTAL/AlphaBetaTHREAD_NUM+1; k<(tid*L_TOTAL/AlphaBetaTHREAD_NUM+L_TOTAL/AlphaBetaTHREAD_NUM); k++) {
        for (s2=0;s2<NSTATE;s2++){
            sum = 0.0;
            for (s1=0;s1<NSTATE;s1++) {
                sum+=exp(gamma[k-1][s2][s1]+Alpha[k-1][s1]);
			}
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                Alpha[k][s2]=-INIFINITY;
            else
                Alpha[k][s2]=log(sum);
        }

		// normalization,prevent overflow
		maxBranch[k]=Alpha[k][0];
		for (s2=1;s2<NSTATE;s2++)
			if (Alpha[k][s2]>maxBranch[k])
				maxBranch[k]=Alpha[k][s2];

		for (s2=0;s2<NSTATE;s2++)
			Alpha[k][s2]=Alpha[k][s2]-maxBranch[k];
	}

}

__global__ void Beta(double (*Beta)[8], double (*gamma)[8][8], bool index, double* maxBranch) {
	const int tid = threadIdx.x;

	UINT k, s1, s2;
	double sum;

	if (tid == (AlphaBetaTHREAD_NUM-1)) {
		if (index){// true -- terminated,false -- open
        Beta[L_TOTAL][0]=0.0;
        for (s2=1;s2<NSTATE;s2++)
            Beta[L_TOTAL][s2]=-INIFINITY;
		}
		else 
			for (s2=0;s2<NSTATE;s2++)
				Beta[L_TOTAL][s2]=0;
	}
	else {
		for (s2=0; s2<NSTATE; s2++)
			Beta[(tid+1)*L_TOTAL/AlphaBetaTHREAD_NUM][s2]=0;
	}

    for (k=(tid+1)*L_TOTAL/AlphaBetaTHREAD_NUM-1;k>(tid*L_TOTAL/AlphaBetaTHREAD_NUM);k--) {
   // for (k=L_TOTAL-1;k>0;k--) {

        for (s1=0;s1<NSTATE;s1++) {
            sum = 0.0;
            for (s2=0;s2<NSTATE;s2++) 
                sum += exp(gamma[k][s1][s2] + Beta[k+1][s2]);
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                Beta[k][s1] = -INIFINITY;
            else 
                Beta[k][s1] = log(sum);
        }

		// normalization,prevent overflow
		for (s2=0;s2<NSTATE;s2++)
			Beta[k][s2]=Beta[k][s2]-maxBranch[k];
	}
}

void computeAlpha(double (*AlphaHost)[8], double (*gamma)[8][8], double *maxBranch) {
    // initialize Alpha & Beta
    AlphaHost[0][0]=0;
	UINT s1,k,s2;
	double sum;
    for (s1=1;s1<NSTATE;s1++)
        AlphaHost[0][s1]=-INIFINITY;

    for (k=1;k<=L_TOTAL;k++){

        for (s2=0;s2<NSTATE;s2++){
            sum = 0;
            for (s1=0;s1<NSTATE;s1++) {
                sum+=exp(gamma[k-1][s2][s1]+AlphaHost[k-1][s1]);
			}
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                AlphaHost[k][s2]=-INIFINITY;
            else
                AlphaHost[k][s2]=log(sum);
        }

		// normalization,prevent overflow
		maxBranch[k]=AlphaHost[k][0];
		for (s2=1;s2<NSTATE;s2++)
			if (AlphaHost[k][s2]>maxBranch[k])
				maxBranch[k]=AlphaHost[k][s2];

		for (s2=0;s2<NSTATE;s2++)
			AlphaHost[k][s2]=AlphaHost[k][s2]-maxBranch[k];
    }
}

void computeBeta(double (*BetaHost)[8], double (*gamma)[8][8], bool index, double * maxBranch){
    // initialize Beta
	UINT s1,k,s2;
	double sum;
    if (index){// true -- terminated,false -- open
        BetaHost[L_TOTAL][0]=0;
        for (s2=1;s2<NSTATE;s2++)
            BetaHost[L_TOTAL][s2]=-INIFINITY;
    }
    else 
        for (s2=0;s2<NSTATE;s2++)
            BetaHost[L_TOTAL][s2]=0;

    for (k=L_TOTAL-1;k>0;k--) {

        for (s1=0;s1<NSTATE;s1++) {
            sum = 0.0;
            for (s2=0;s2<NSTATE;s2++) 
                sum += exp(gamma[k][s1][s2] + BetaHost[k+1][s2]);
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                BetaHost[k][s1] = -INIFINITY;
            else 
                BetaHost[k][s1] = log(sum);
        }

		// normalization,prevent overflow
		for (s2=0;s2<NSTATE;s2++)
			BetaHost[k][s2]=BetaHost[k][s2]-maxBranch[k];
    }
}

__global__ void normalizationAlphaAndBeta(double (*Alpha)[8], double (*Beta)[8]) {
    unsigned int tid = threadIdx.x+1; 
    double max_branch;
    max_branch = Alpha[tid][0];
	UINT s2;
    for (s2=1;s2<NSTATE;s2++)
        if (Alpha[tid][s2]>max_branch)
            max_branch = Alpha[tid][s2];

    for (s2=0;s2<NSTATE;s2++) {
        Alpha[tid][s2] = Alpha[tid][s2] - max_branch;

        if (tid != L_TOTAL) 
            Beta[tid][s2] = Beta[tid][s2] - max_branch;
    }

}

__global__ void LLRS(double * msg, double * parity, double * L_a, double (*Alpha)[8], double (*Beta)[8], double * L_all,BYTE (*lastState)[8], char (*lastOut)[8]) {
    unsigned int tid = threadIdx.x; 
    UINT s2;
	double sum0 = 0.0, sum1 = 0.0;
    for (s2=0;s2<NSTATE;s2++) {
        //gamma[LastState[0][s2]]=-msg[tid]+parity[tid]*LastOut[0][s2]-log(1+exp(L_a[tid]));
        //gamma[LastState[1][s2]]=msg[tid]+parity[tid]*LastOut[1][s2]+L_a[tid]-log(1+exp(L_a[tid]));
        //sum0+=exp(gamma[LastState[0][s2]]+Alpha[tid][LastState[0][s2]]+Beta[tid+1][s2]);
        //sum1+=exp(gamma[LastState[1][s2]]+Alpha[tid][LastState[1][s2]]+Beta[tid+1][s2]);
        double gamma0=-msg[tid]+parity[tid]*lastOut[0][s2]-log(1+exp(L_a[tid]));
        double gamma1=msg[tid]+parity[tid]*lastOut[1][s2]+L_a[tid]-log(1+exp(L_a[tid]));
        sum0+=exp(gamma0+Alpha[tid][lastState[0][s2]]+Beta[tid+1][s2]);
        sum1+=exp(gamma1+Alpha[tid][lastState[1][s2]]+Beta[tid+1][s2]);
    }
    //L_all[tid]=log(sum1)-log(sum0);
    L_all[tid]=log(sum1)-log(sum0);
}

__global__ void extrinsicInformation(double * L_all, double * msg, double * L_a, double * L_e) {
    unsigned int tid = threadIdx.x;
    L_e[tid] = L_all[tid] - 2*msg[tid] - L_a[tid];
}

__global__ void demultiplex(double * stream, double * msg, double * parity0, double * parity1) {
    unsigned int tid = threadIdx.x;
    //if (puncture){// punctured rate=1/2
    //    msg[tid]=stream[2*tid];
    //    parity[tid%2][tid]=stream[tid*2+1];
    //}
    //else {// unpunctured rate=1/3
    //    msg[tid]=stream[3*tid];
    //    parity0[tid]=stream[3*tid+1];
    //    parity1[tid]=stream[3*tid+2];
    //}
        msg[tid]=stream[3*tid];
        parity0[tid]=stream[3*tid+1];
        parity1[tid]=stream[3*tid+2];
}

__global__ void initializeExtrinsicInformation(double * L_e) {
    unsigned int tid = threadIdx.x;
    L_e[tid] = 0;
    
}

__global__ void exestimateInformationBits(double * L_all, BYTE * msghat, UINT * interLeaveTable[3]) {
//__global__ void exestimateInformationBits(double * L_all, BYTE * msghat, UINT * m_Inter_table) {
    unsigned int tid = threadIdx.x;
    if(L_all[tid]>0)
		msghat[(interLeaveTable[tid][1]*tid+interLeaveTable[tid][2]*tid*tid)%interLeaveTable[tid][0]]=1;
        //msghat[m_Inter_table[tid]]=1;
    else
		msghat[(interLeaveTable[tid][1]*tid+interLeaveTable[tid][2]*tid*tid)%interLeaveTable[tid][0]]=0;
        //msghat[m_Inter_table[tid]]=0;
}

void countErrors(BYTE *m, BYTE * mhat, UINT * bitsError, UINT * frameError, UINT iter) {

	bool f_err = false;
	for (int i=0; i<(L_TOTAL-M);i++) {
		if (m[i] != mhat[i]) {
			bitsError[iter] = bitsError[iter]+1;
			f_err = true;
		}
	}

	if (f_err) 
		frameError[iter] = frameError[iter]+1;
}


int main(int argc, char* argv[])
{
    initRandom(seed);

	BYTE * m;
	BYTE * x;
	double * y;
	BYTE * mhat;

	int frame;
	UINT bits_all,bits_err[MAXITER],frame_err[MAXITER];
	double Ber,Fer;
	double Eb_No_dB,No;
	//bool f_err;
	//FILE * fp;
	int i;

	m = new BYTE[L_TOTAL];
	x = new BYTE[L_ALL];
	y = new double[L_ALL];
	mhat = new BYTE[L_TOTAL];

	//init_Block_interleave_table();	// block interleave
	//init_my_interleave_table();


    
    findCudaDevice(argc, (const char **)argv);

	BYTE (*LastStateDevice)[8];
	BYTE (*NextStateDevice)[8];
	char (*LastOutDevice)[8];
	char (*NextOutDevice)[8];
	double * yDevice;
	double * msgDevice;
	double * imsgDevice;
	BYTE * mhatDevice;
	double * parity0Device;
	double * parity1Device;
	//UINT * tableDevice;
	UINT * LTETableDevice[3];
	double * L_eDevice;
	double * L_aDevice;
	double * L_allDevice;
	double (*gammaAlphaDevice)[8][8];
	double (*gammaBetaDevice)[8][8];
	double (*AlphaDevice)[8];
	double (*BetaDevice)[8];
	double *maxBranchDevice;

    hipMalloc((void **)&LastStateDevice, 2*8*sizeof(BYTE));
    hipMalloc((void **)&NextStateDevice, 2*8*sizeof(BYTE));
    hipMalloc((void **)&LastOutDevice, 2*8*sizeof(char));
    hipMalloc((void **)&NextOutDevice, 2*8*sizeof(char));

    hipMalloc((void **)&yDevice, L_ALL*sizeof(double));
    hipMalloc((void **)&msgDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&imsgDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&mhatDevice, L_TOTAL*sizeof(BYTE));
    hipMalloc((void **)&parity0Device, L_TOTAL*sizeof(double));
    hipMalloc((void **)&parity1Device, L_TOTAL*sizeof(double));
    //hipMalloc((void **)&tableDevice, L_TOTAL*sizeof(unsigned int));
    hipMalloc((void **)&LTETableDevice, L_TOTAL*3*sizeof(unsigned int));
    hipMalloc((void **)&L_eDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&L_aDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&L_allDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&gammaAlphaDevice, L_TOTAL*sizeof(double)*8*8);
    hipMalloc((void **)&gammaBetaDevice, L_TOTAL*sizeof(double)*8*8);
    hipMalloc((void **)&AlphaDevice, (L_TOTAL+1)*sizeof(double)*8);
    hipMalloc((void **)&BetaDevice, (L_TOTAL+1)*sizeof(double)*8);
    hipMalloc((void **)&maxBranchDevice, (L_TOTAL+1)*sizeof(double));

	//For Debug
	//double L_aHost[L_TOTAL];
	//double L_aHost1[L_TOTAL];
	//double L_allHost[L_TOTAL];

    double gammaAlphaHost[L_TOTAL][8][8];
    double gammaBetaHost[L_TOTAL][8][8];
    double AlphaHost[L_TOTAL+1][8];
    double BetaHost[L_TOTAL+1][8];

	double max_branch[L_TOTAL+1];

    hipMemcpy(LastStateDevice,LastState,sizeof(BYTE)*2*8, hipMemcpyHostToDevice);
    hipMemcpy(NextStateDevice,NextState,sizeof(BYTE)*2*8, hipMemcpyHostToDevice);
    hipMemcpy(LastOutDevice,LastOut,sizeof(char)*2*8, hipMemcpyHostToDevice);
    hipMemcpy(NextOutDevice,NextOut,sizeof(char)*2*8, hipMemcpyHostToDevice);

    //hipMemcpy(tableDevice,m_Inter_table,sizeof(unsigned int)*L_TOTAL, hipMemcpyHostToDevice);
    hipMemcpy(LTETableDevice,LTE_Inter_table,sizeof(unsigned int)*L_TOTAL*3, hipMemcpyHostToDevice);

	for (Eb_No_dB= 0.0;Eb_No_dB<1.0;Eb_No_dB+=1){

	//Eb_No_dB = 0.0;
		No = 1/pow(10.0,Eb_No_dB/10.0);
		bits_all = 0;
		for (i =0; i<MAXITER;i++) {
			bits_err[i]=0;
			frame_err[i]=0;
		}

		for (frame = 0; frame<FRAME_NUM; frame++, bits_all += (L_TOTAL-M)) {

			// Generate random information bits
			for (i=0;i<L_TOTAL;i++)
				if (boolrandom())
					m[i]=1;
				else
					m[i]=0;
			// encoder
			encode(m,x,false);
			// add noise
			for (i=0;i<L_ALL;i++)
				if (x[i])
					y[i]=1.0+gaussian(No/2);
				else
					y[i]=-1.0+gaussian(No/2);

			hipMemcpy(yDevice,y,sizeof(double)*L_ALL, hipMemcpyHostToDevice);

			demultiplex<<<1,L_TOTAL>>>(yDevice, msgDevice, parity0Device, parity1Device); 
			//interLeave<<<1,L_TOTAL>>>(msgDevice, imsgDevice, tableDevice);
			interLeave<<<1,L_TOTAL>>>(msgDevice, imsgDevice, LTETableDevice);
			initializeExtrinsicInformation<<<1,L_TOTAL>>>(L_eDevice);

			for (int iter = 0; iter<MAXITER; iter++) {
				
			//	deInterLeave<<<1,L_TOTAL>>>(L_eDevice, L_aDevice, tableDevice);
				deInterLeave<<<1,L_TOTAL>>>(L_eDevice, L_aDevice, LTETableDevice);

				gammaAlpha<<<1,L_TOTAL>>>(msgDevice , parity0Device,  L_aDevice,  gammaAlphaDevice,LastStateDevice, LastOutDevice);
				gammaBeta<<<1,L_TOTAL>>>(msgDevice , parity0Device,  L_aDevice,  gammaBetaDevice, NextStateDevice, NextOutDevice);
				Alpha<<<1,AlphaBetaTHREAD_NUM>>>(AlphaDevice, gammaAlphaDevice, maxBranchDevice);
				Beta<<<1,AlphaBetaTHREAD_NUM>>>(BetaDevice, gammaBetaDevice,true, maxBranchDevice);
				hipMemcpy(AlphaHost, AlphaDevice, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyDeviceToHost);
				hipMemcpy(gammaAlphaHost, gammaAlphaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);
				//hipMemcpy(gammaBetaHost, gammaBetaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);

				//computeAlpha(AlphaHost, gammaAlphaHost, max_branch);
				//computeBeta(BetaHost, gammaBetaHost, true,max_branch);
				//hipMemcpy(AlphaDevice, AlphaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				//hipMemcpy(BetaDevice, BetaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				normalizationAlphaAndBeta<<<1,L_TOTAL>>>(AlphaDevice, BetaDevice);
				hipMemcpy(AlphaHost, AlphaDevice, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyDeviceToHost);

				LLRS<<<1,L_TOTAL>>>(msgDevice, parity0Device, L_aDevice, AlphaDevice, BetaDevice, L_allDevice,LastStateDevice, LastOutDevice);

				extrinsicInformation<<<1, L_TOTAL>>>(L_allDevice, msgDevice, L_aDevice, L_eDevice);
				//if (iter >= 3) {
				///debug
				//exestimateInformationBits<<<1,L_TOTAL>>>(L_allDevice, mhatDevice, tableDevice); 

				//hipMemcpy(mhat, mhatDevice, sizeof(BYTE)*L_TOTAL, hipMemcpyDeviceToHost);
				//countErrors(m, mhat, bits_err, frame_err, iter);
				//debug
				//hipMemcpy(L_aHost1, L_aDevice, sizeof(double)*L_TOTAL, hipMemcpyDeviceToHost);
				//}

				//interLeave<<<1, L_TOTAL>>>(L_eDevice, L_aDevice, tableDevice);
				interLeave<<<1, L_TOTAL>>>(L_eDevice, L_aDevice, LTETableDevice);

				gammaAlpha<<<1,L_TOTAL>>>(imsgDevice , parity1Device,  L_aDevice,  gammaAlphaDevice, LastStateDevice, LastOutDevice);
				gammaBeta<<<1,L_TOTAL>>>(imsgDevice , parity1Device,  L_aDevice,  gammaBetaDevice, NextStateDevice, NextOutDevice);
				Alpha<<<1,AlphaBetaTHREAD_NUM>>>(AlphaDevice, gammaAlphaDevice, maxBranchDevice);
				Beta<<<1,AlphaBetaTHREAD_NUM>>>(BetaDevice, gammaBetaDevice,false, maxBranchDevice);
				hipMemcpy(AlphaHost, AlphaDevice, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyDeviceToHost);
				//hipMemcpy(gammaAlphaHost, gammaAlphaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);
				//hipMemcpy(gammaBetaHost, gammaBetaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);

				//computeAlpha(AlphaHost, gammaAlphaHost, max_branch);
				//Beta(BetaHost, gammaBetaHost, false, max_branch);
				//hipMemcpy(AlphaDevice, AlphaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				//hipMemcpy(BetaDevice, BetaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				normalizationAlphaAndBeta<<<1,L_TOTAL>>>(AlphaDevice, BetaDevice);
				hipMemcpy(AlphaHost, AlphaDevice, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyDeviceToHost);

				LLRS<<<1,L_TOTAL>>>(imsgDevice, parity1Device, L_aDevice, AlphaDevice, BetaDevice, L_allDevice, LastStateDevice,LastOutDevice);

				extrinsicInformation<<<1, L_TOTAL>>>(L_allDevice, imsgDevice, L_aDevice, L_eDevice);

				exestimateInformationBits<<<1,L_TOTAL>>>(L_allDevice, mhatDevice, LTETableDevice); 
				//exestimateInformationBits<<<1,L_TOTAL>>>(L_allDevice, mhatDevice, tableDevice); 

				hipMemcpy(mhat, mhatDevice, sizeof(BYTE)*L_TOTAL, hipMemcpyDeviceToHost);
				countErrors(m, mhat, bits_err, frame_err, iter);

				//debug
				//hipMemcpy(L_aHost, L_aDevice, sizeof(double)*L_TOTAL, hipMemcpyDeviceToHost);
			}
			// estimate information bits
			//exestimateInformationBits<<<1,L_TOTAL>>>(L_allDevice, mhatDevice, tableDevice); 

			//hipMemcpy(mhat, mhatDevice, sizeof(BYTE)*L_TOTAL, hipMemcpyDeviceToHost);
			// count errors
			//UINT bits_err = 0;
			//for (i=0;i<L_TOTAL-M;i++) {
			//	if (mhat[i]!=m[i]) {
			//		bits_err++;
			//	}
			//}
			//cout<<"bits_err: "<<bits_err<<endl;
		}

		printf("-------------------------\n");
		printf("Eb/No=%fdB:\n",Eb_No_dB);
		printf("-------------------------\n");
		//fprintf(fp,"-------------------------\n");
		//fprintf(fp,"Eb/No=%fdB:\n",Eb_No_dB);
		//fprintf(fp,"-------------------------\n");

		for (i=0;i<MAXITER;i++) {
			Ber=(double)bits_err[i]/(double)bits_all;
			Fer=(double)frame_err[i]/(double)FRAME_NUM;
			printf("Iteration:%d\n",i+1);
			printf("---Ber=%f\n---Fer=%f\n",Ber,Fer);
			//fprintf(fp,"Iteration:%d\n",i);
			//fprintf(fp,"---Ber=%f\n---Fer=%f\n",Ber,Fer);
		}
	}

	delete m;
	delete x;
	delete y;
	delete mhat;

	hipFree(LastStateDevice);
	hipFree(NextStateDevice);
	hipFree(LastOutDevice);
	hipFree(NextOutDevice);
	hipFree(yDevice);
	hipFree(msgDevice);
	hipFree(imsgDevice);
	hipFree(mhatDevice);
	hipFree(parity0Device);
	hipFree(parity1Device);
//	hipFree(tableDevice);
	hipFree(LTETableDevice);
	hipFree(L_eDevice);
	hipFree(L_aDevice);
	hipFree(L_allDevice);
	hipFree(gammaAlphaDevice);
	hipFree(gammaBetaDevice);
	hipFree(AlphaDevice);
	hipFree(BetaDevice);

	return 0;
}
