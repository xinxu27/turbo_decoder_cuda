#include "hip/hip_runtime.h"
// TurboDecoder : Defines the entry point for the console application.
#include "hip/hip_runtime_api.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <limits.h>
#include <iostream>
using namespace std;


#define IM1 2147483563
#define IM2 2147483399
#define AM (1.0/IM1)
#define IMM1 (IM1-1)
#define IA1 40014
#define IA2 40692
#define IQ1 53668
#define IQ2 52774
#define IR1 12211
#define IR2 3791
#define NTAB 32
#define NDIV (1+IMM1/NTAB)
#define EPS 1.7E-308 //MINDOUBLE
#define RNMX (1.0-EPS)
#define INIFINITY  1E+10

#define MIN 1E-300
#define L_TOTAL 4096// if u want to use block interleave,L_TOTAL must = x^2
#define MAXITER 10
#define	FRAME_NUM 10
#define AlphaBetaTHREAD_NUM 4

#define THREAD_NUM 1024
#define BLOCK_NUM 4

//typedef enum __bool { false = 0, true = 1, } bool;

long idum2;
long idum;
long iy;
long iv[NTAB];	
unsigned memory;

/*
Long period (? 2 \Theta 10 18 ) random number generator of L'Ecuyer with Bays­Durham shuffle
and added safeguards. Returns a uniform random deviate between 0.0 and 1.0 (exclusive of
the endpoint values). Call with idum a negative integer to initialize; thereafter, do not alter
idum between successive deviates in a sequence. RNMX should approximate the largest floating
value that is less than 1.
--
*/


double ran2()
{
	int j;
	long k;
	double temp;
	
	
	k=(idum)/IQ1;
	idum=IA1*(idum-k*IQ1)-k*IR1;  // Compute idum=(IA1*idum) % IM1 without overflows by Schrage's method.
	if (idum < 0)
		idum += IM1;
	k=idum2/IQ2;
	idum2=IA2*(idum2-k*IQ2)-k*IR2;	// Compute idum2=(IA2*idum) % IM2 likewise.
	if (idum2 < 0)
		idum2 += IM2;
	j = iy/NDIV;
	iy=iv[j]-idum2;
	//	iy=iv[j]-idum2; 		// Here idum is shuffled, idum and idum2 are combined to generate output.
	iv[j] = idum;
	if (iy < 1)
		iy += IMM1;
	if ((temp=AM*iy) > RNMX)
		return RNMX; 		// Because users don't expect endpoint values.
	else
		return temp;
}


void initRandom(long seed)
{
	idum2=123456789;
	idum=0;
	iy=0;
	
	if (seed != 0)
		idum = seed;
	else
		idum = 1;
	
	
    int j;
	for (j=NTAB+7;j>=0;j--) // Load the shuffle table (after 8 warm­ups).
	{
		long k=(idum)/IQ1;
		
		idum=IA1*(idum-k*IQ1)-k*IR1;
		if (idum < 0)
			idum += IM1;
		if (j < NTAB)
			iv[j] = idum;
	}
	iy=iv[0];
}


double doublerandom()
{
	double t = ran2();
	return t;
}

long longrandom(long range)
{
	double t;
	
	t = doublerandom();
	return((long)(t*(double)range));
}


bool boolrandom()
{
	double t=doublerandom();
	
	if (t>0.5)
		return true;
	else
		return false;
}
//time_t t;	
//time(&t);	
//init((long)t);
long seed = 1234421;




#define M	3	// register length,=tail length
#define NSTATE	8	// = M^2
#define L_ALL 3*L_TOTAL	// coded frame length
//#define DELTA 30	// SOVA window size. Make decision after 'delta' delay. Decide bit k when received bits
					// for bit (k+delta) are processed. Trace back from (k+delta) to k. 
// Rate 1/3 Turbo code
// The generator polynomials used are:
//	g0=[1 1 1]
//  g1=[1 0 1]
//	RSC encoder structure:
//
//			 +------------------------------------------>c1
//           |          g0(x)    
//           x_.--o-------------(+)<----------+
//           |    |             /|\           |
//			 |   \|/      ---    |     ---    |
// data --_.-o-->(+)--o->| D0|---o--->| D1|---o
//                    |   ---          ---    |
//                    |                       |
//                    +-------->(+)<----------+
//                      g1(x)    |
//								 +---------------------->c2
//
typedef unsigned char BYTE;
typedef int INT;
typedef unsigned int UINT;
typedef int BOOL;

// NextOut[bk][current state]
static const char EnNextOut[2][NSTATE] = // check bit based on current and input bit
{	0,0,1,1,1,1,0,0,
	0,0,1,1,1,1,0,0
};
static const char NextOut[2][NSTATE] = // check bit based on current and input bit
{	-1,-1,1,1,1,1,-1,-1,
	-1,-1,1,1,1,1,-1,-1
};
// NextState[bk][current state]
static const BYTE NextState[2][NSTATE] = // next state based on current and input bit
{	0,4,5,1,2,6,7,3,
	4,0,1,5,6,2,3,7
};
// LastOut[bk][current state]
static const char LastOut[2][NSTATE] =	// trellis last check bit
{	-1,1,1,-1,-1,1,1,-1,
	-1,1,1,-1,-1,1,1,-1
};
// LastState[bk][current state]
static const BYTE LastState[2][NSTATE] =	// last state lead to current state by input bk
{	0,3,4,7,1,2,5,6,
	1,2,5,6,0,3,4,7
};
// TailBit[current state]
static const char TailBit[NSTATE] = // tail info bits when trellis is terminating
{	0,1,1,0,0,1,1,0
};



UINT m_Inter_table[L_TOTAL] = 
{0,95,318,669,1148,1755,2490,3353,248,1367,
2614,3989,1396,3027,690,2577,496,2639,814,3213,
1644,203,2986,1801,744,3911,3110,2437,1892,1475,
1186,1025,992,1087,1310,1661,2140,2747,3482,249,
1240,2359,3606,885,2388,4019,1682,3569,1488,3631,
1806,109,2636,1195,3978,2793,1736,807,6,3429,
2884,2467,2178,2017,1984,2079,2302,2653,3132,3739,
378,1241,2232,3351,502,1877,3380,915,2674,465,
2480,527,2798,1101,3628,2187,874,3785,2728,1799,
998,325,3876,3459,3170,3009,2976,3071,3294,3645,
28,635,1370,2233,3224,247,1494,2869,276,1907,
3666,1457,3472,1519,3790,2093,524,3179,1866,681,
3720,2791,1990,1317,772,355,66,4001,3968,4063,
190,541,1020,1627,2362,3225,120,1239,2486,3861,
1268,2899,562,2449,368,2511,686,3085,1516,75,
2858,1673,616,3783,2982,2309,1764,1347,1058,897,
864,959,1182,1533,2012,2619,3354,121,1112,2231,
3478,757,2260,3891,1554,3441,1360,3503,1678,4077,
2508,1067,3850,2665,1608,679,3974,3301,2756,2339,
2050,1889,1856,1951,2174,2525,3004,3611,250,1113,
2104,3223,374,1749,3252,787,2546,337,2352,399,
2670,973,3500,2059,746,3657,2600,1671,870,197,
3748,3331,3042,2881,2848,2943,3166,3517,3996,507,
1242,2105,3096,119,1366,2741,148,1779,3538,1329,
3344,1391,3662,1965,396,3051,1738,553,3592,2663,
1862,1189,644,227,4034,3873,3840,3935,62,413,
892,1499,2234,3097,4088,1111,2358,3733,1140,2771,
434,2321,240,2383,558,2957,1388,4043,2730,1545,
488,3655,2854,2181,1636,1219,930,769,736,831,
1054,1405,1884,2491,3226,4089,984,2103,3350,629,
2132,3763,1426,3313,1232,3375,1550,3949,2380,939,
3722,2537,1480,551,3846,3173,2628,2211,1922,1761,
1728,1823,2046,2397,2876,3483,122,985,1976,3095,
246,1621,3124,659,2418,209,2224,271,2542,845,
3372,1931,618,3529,2472,1543,742,69,3620,3203,
2914,2753,2720,2815,3038,3389,3868,379,1114,1977,
2968,4087,1238,2613,20,1651,3410,1201,3216,1263,
3534,1837,268,2923,1610,425,3464,2535,1734,1061,
516,99,3906,3745,3712,3807,4030,285,764,1371,
2106,2969,3960,983,2230,3605,1012,2643,306,2193,
112,2255,430,2829,1260,3915,2602,1417,360,3527,
2726,2053,1508,1091,802,641,608,703,926,1277,
1756,2363,3098,3961,856,1975,3222,501,2004,3635,
1298,3185,1104,3247,1422,3821,2252,811,3594,2409,
1352,423,3718,3045,2500,2083,1794,1633,1600,1695,
1918,2269,2748,3355,4090,857,1848,2967,118,1493,
2996,531,2290,81,2096,143,2414,717,3244,1803,
490,3401,2344,1415,614,4037,3492,3075,2786,2625,
2592,2687,2910,3261,3740,251,986,1849,2840,3959,
1110,2485,3988,1523,3282,1073,3088,1135,3406,1709,
140,2795,1482,297,3336,2407,1606,933,388,4067,
3778,3617,3584,3679,3902,157,636,1243,1978,2841,
3832,855,2102,3477,884,2515,178,2065,4080,2127,
302,2701,1132,3787,2474,1289,232,3399,2598,1925,
1380,963,674,513,480,575,798,1149,1628,2235,
2970,3833,728,1847,3094,373,1876,3507,1170,3057,
976,3119,1294,3693,2124,683,3466,2281,1224,295,
3590,2917,2372,1955,1666,1505,1472,1567,1790,2141,
2620,3227,3962,729,1720,2839,4086,1365,2868,403,
2162,4049,1968,15,2286,589,3116,1675,362,3273,
2216,1287,486,3909,3364,2947,2658,2497,2464,2559,
2782,3133,3612,123,858,1721,2712,3831,982,2357,
3860,1395,3154,945,2960,1007,3278,1581,12,2667,
1354,169,3208,2279,1478,805,260,3939,3650,3489,
3456,3551,3774,29,508,1115,1850,2713,3704,727,
1974,3349,756,2387,50,1937,3952,1999,174,2573,
1004,3659,2346,1161,104,3271,2470,1797,1252,835,
546,385,352,447,670,1021,1500,2107,2842,3705,
600,1719,2966,245,1748,3379,1042,2929,848,2991,
1166,3565,1996,555,3338,2153,1096,167,3462,2789,
2244,1827,1538,1377,1344,1439,1662,2013,2492,3099,
3834,601,1592,2711,3958,1237,2740,275,2034,3921,
1840,3983,2158,461,2988,1547,234,3145,2088,1159,
358,3781,3236,2819,2530,2369,2336,2431,2654,3005,
3484,4091,730,1593,2584,3703,854,2229,3732,1267,
3026,817,2832,879,3150,1453,3980,2539,1226,41,
3080,2151,1350,677,132,3811,3522,3361,3328,3423,
3646,3997,380,987,1722,2585,3576,599,1846,3221,
628,2259,4018,1809,3824,1871,46,2445,876,3531,
2218,1033,4072,3143,2342,1669,1124,707,418,257,
224,319,542,893,1372,1979,2714,3577,472,1591,
2838,117,1620,3251,914,2801,720,2863,1038,3437,
1868,427,3210,2025,968,39,3334,2661,2116,1699,
1410,1249,1216,1311,1534,1885,2364,2971,3706,473,
1464,2583,3830,1109,2612,147,1906,3793,1712,3855,
2030,333,2860,1419,106,3017,1960,1031,230,3653,
3108,2691,2402,2241,2208,2303,2526,2877,3356,3963,
602,1465,2456,3575,726,2101,3604,1139,2898,689,
2704,751,3022,1325,3852,2411,1098,4009,2952,2023,
1222,549,4,3683,3394,3233,3200,3295,3518,3869,
252,859,1594,2457,3448,471,1718,3093,500,2131,
3890,1681,3696,1743,4014,2317,748,3403,2090,905,
3944,3015,2214,1541,996,579,290,129,96,191,
414,765,1244,1851,2586,3449,344,1463,2710,4085,
1492,3123,786,2673,592,2735,910,3309,1740,299,
3082,1897,840,4007,3206,2533,1988,1571,1282,1121,
1088,1183,1406,1757,2236,2843,3578,345,1336,2455,
3702,981,2484,19,1778,3665,1584,3727,1902,205,
2732,1291,4074,2889,1832,903,102,3525,2980,2563,
2274,2113,2080,2175,2398,2749,3228,3835,474,1337,
2328,3447,598,1973,3476,1011,2770,561,2576,623,
2894,1197,3724,2283,970,3881,2824,1895,1094,421,
3972,3555,3266,3105,3072,3167,3390,3741,124,731,
1466,2329,3320,343,1590,2965,372,2003,3762,1553,
3568,1615,3886,2189,620,3275,1962,777,3816,2887,
2086,1413,868,451,162,1,4064,63,286,637,
1116,1723,2458,3321,216,1335,2582,3957,1364,2995,
658,2545,464,2607,782,3181,1612,171,2954,1769,
712,3879,3078,2405,1860,1443,1154,993,960,1055,
1278,1629,2108,2715,3450,217,1208,2327,3574,853,
2356,3987,1650,3537,1456,3599,1774,77,2604,1163,
3946,2761,1704,775,4070,3397,2852,2435,2146,1985,
1952,2047,2270,2621,3100,3707,346,1209,2200,3319,
470,1845,3348,883,2642,433,2448,495,2766,1069,
3596,2155,842,3753,2696,1767,966,293,3844,3427,
3138,2977,2944,3039,3262,3613,4092,603,1338,2201,
3192,215,1462,2837,244,1875,3634,1425,3440,1487,
3758,2061,492,3147,1834,649,3688,2759,1958,1285,
740,323,34,3969,3936,4031,158,509,988,1595,
2330,3193,88,1207,2454,3829,1236,2867,530,2417,
336,2479,654,3053,1484,43,2826,1641,584,3751,
2950,2277,1732,1315,1026,865,832,927,1150,1501,
1980,2587,3322,89,1080,2199,3446,725,2228,3859,
1522,3409,1328,3471,1646,4045,2476,1035,3818,2633,
1576,647,3942,3269,2724,2307,2018,1857,1824,1919,
2142,2493,2972,3579,218,1081,2072,3191,342,1717,
3220,755,2514,305,2320,367,2638,941,3468,2027,
714,3625,2568,1639,838,165,3716,3299,3010,2849,
2816,2911,3134,3485,3964,475,1210,2073,3064,87,
1334,2709,116,1747,3506,1297,3312,1359,3630,1933,
364,3019,1706,521,3560,2631,1830,1157,612,195,
4002,3841,3808,3903,30,381,860,1467,2202,3065,
4056,1079,2326,3701,1108,2739,402,2289,208,2351,
526,2925,1356,4011,2698,1513,456,3623,2822,2149,
1604,1187,898,737,704,799,1022,1373,1852,2459,
3194,4057,952,2071,3318,597,2100,3731,1394,3281,
1200,3343,1518,3917,2348,907,3690,2505,1448,519,
3814,3141,2596,2179,1890,1729,1696,1791,2014,2365,
2844,3451,90,953,1944,3063,214,1589,3092,627,
2386,177,2192,239,2510,813,3340,1899,586,3497,
2440,1511,710,37,3588,3171,2882,2721,2688,2783,
3006,3357,3836,347,1082,1945,2936,4055,1206,2581,
4084,1619,3378,1169,3184,1231,3502,1805,236,2891,
1578,393,3432,2503,1702,1029,484,67,3874,3713,
3680,3775,3998,253,732,1339,2074,2937,3928,951,
2198,3573,980,2611,274,2161,80,2223,398,2797,
1228,3883,2570,1385,328,3495,2694,2021,1476,1059,
770,609,576,671,894,1245,1724,2331,3066,3929,
824,1943,3190,469,1972,3603,1266,3153,1072,3215,
1390,3789,2220,779,3562,2377,1320,391,3686,3013,
2468,2051,1762,1601,1568,1663,1886,2237,2716,3323,
4058,825,1816,2935,86,1461,2964,499,2258,49,
2064,111,2382,685,3212,1771,458,3369,2312,1383,
582,4005,3460,3043,2754,2593,2560,2655,2878,3229,
3708,219,954,1817,2808,3927,1078,2453,3956,1491,
3250,1041,3056,1103,3374,1677,108,2763,1450,265,
3304,2375,1574,901,356,4035,3746,3585,3552,3647,
3870,125,604,1211,1946,2809,3800,823,2070,3445,
852,2483,146,2033,4048,2095,270,2669,1100,3755,
2442,1257,200,3367,2566,1893,1348,931,642,481,
448,543,766,1117,1596,2203,2938,3801,696,1815,
3062,341,1844,3475,1138,3025,944,3087,1262,3661,
2092,651,3434,2249,1192,263,3558,2885,2340,1923,
1634,1473,1440,1535,1758,2109,2588,3195,3930,697,
1688,2807,4054,1333,2836,371,2130,4017,1936,4079,
2254,557,3084,1643,330,3241,2184,1255,454,3877,
3332,2915,2626,2465,2432,2527,2750,3101,3580,91,
826,1689,2680,3799,950,2325,3828,1363,3122,913,
2928,975,3246,1549,4076,2635,1322,137,3176,2247,
1446,773,228,3907,3618,3457,3424,3519,3742,4093,
476,1083,1818,2681,3672,695,1942,3317,724,2355,
18,1905,3920,1967,142,2541,972,3627,2314,1129,
72,3239,2438,1765,1220,803,514,353,320,415,
638,989,1468,2075,2810,3673,568,1687,2934,213,
1716,3347,1010,2897,816,2959,1134,3533,1964,523,
3306,2121,1064,135,3430,2757,2212,1795,1506,1345,
1312,1407,1630,1981,2460,3067,3802,569,1560,2679,
3926,1205,2708,243,2002,3889,1808,3951,2126,429,
2956,1515,202,3113,2056,1127,326,3749,3204,2787,
2498,2337,2304,2399,2622,2973,3452,4059,698,1561,
2552,3671,822,2197,3700,1235,2994,785,2800,847,
3118,1421,3948,2507,1194,9,3048,2119,1318,645,
100,3779,3490,3329,3296,3391,3614,3965,348,955,
1690,2553,3544,567,1814,3189,596,2227,3986,1777,
3792,1839,14,2413,844,3499,2186,1001,4040,3111,
2310,1637,1092,675,386,225,192,287,510,861,
1340,1947,2682,3545,440,1559,2806,85,1588,3219,
882,2769,688,2831,1006,3405,1836,395,3178,1993,
936,7,3302,2629,2084,1667,1378,1217,1184,1279,
1502,1853,2332,2939,3674,441,1432,2551,3798,1077,
2580,115,1874,3761,1680,3823,1998,301,2828,1387,
74,2985,1928,999,198,3621,3076,2659,2370,2209,
2176,2271,2494,2845,3324,3931,570,1433,2424,3543,
694,2069,3572,1107,2866,657,2672,719,2990,1293,
3820,2379,1066,3977,2920,1991,1190,517,4068,3651,
3362,3201,3168,3263,3486,3837,220,827,1562,2425,
3416,439,1686,3061,468,2099,3858,1649,3664,1711,
3982,2285,716,3371,2058,873,3912,2983,2182,1509,
964,547,258,97,64,159,382,733,1212,1819,
2554,3417,312,1431,2678,4053,1460,3091,754,2641,
560,2703,878,3277,1708,267,3050,1865,808,3975,
3174,2501,1956,1539,1250,1089,1056,1151,1374,1725,
2204,2811,3546,313,1304,2423,3670,949,2452,4083,
1746,3633,1552,3695,1870,173,2700,1259,4042,2857,
1800,871,70,3493,2948,2531,2242,2081,2048,2143,
2366,2717,3196,3803,442,1305,2296,3415,566,1941,
3444,979,2738,529,2544,591,2862,1165,3692,2251,
938,3849,2792,1863,1062,389,3940,3523,3234,3073,
3040,3135,3358,3709,92,699,1434,2297,3288,311,
1558,2933,340,1971,3730,1521,3536,1583,3854,2157,
588,3243,1930,745,3784,2855,2054,1381,836,419,
130,4065,4032,31,254,605,1084,1691,2426,3289,
184,1303,2550,3925,1332,2963,626,2513,432,2575,
750,3149,1580,139,2922,1737,680,3847,3046,2373,
1828,1411,1122,961,928,1023,1246,1597,2076,2683,
3418,185,1176,2295,3542,821,2324,3955,1618,3505,
1424,3567,1742,45,2572,1131,3914,2729,1672,743,
4038,3365,2820,2403,2114,1953,1920,2015,2238,2589,
3068,3675,314,1177,2168,3287,438,1813,3316,851,
2610,401,2416,463,2734,1037,3564,2123,810,3721,
2664,1735,934,261,3812,3395,3106,2945,2912,3007,
3230,3581,4060,571,1306,2169,3160,183,1430,2805,
212,1843,3602,1393,3408,1455,3726,2029,460,3115,
1802,617,3656,2727,1926,1253,708,291,2,3937,
3904,3999,126,477,956,1563,2298,3161,56,1175,
2422,3797,1204,2835,498,2385,304,2447,622,3021,
1452,11,2794,1609,552,3719,2918,2245,1700,1283,
994,833,800,895,1118,1469,1948,2555,3290,57,
1048,2167,3414,693,2196,3827,1490,3377,1296,3439,
1614,4013,2444,1003,3786,2601,1544,615,3910,3237,
2692,2275,1986,1825,1792,1887,2110,2461,2940,3547,
186,1049,2040,3159,310,1685,3188,723,2482,273,
2288,335,2606,909,3436,1995,682,3593,2536,1607,
806,133,3684,3267,2978,2817,2784,2879,3102,3453,
3932,443,1178,2041,3032,55,1302,2677,84,1715,
3474,1265,3280,1327,3598,1901,332,2987,1674,489,
3528,2599,1798,1125,580,163,3970,3809,3776,3871,
4094,349,828,1435,2170,3033,4024,1047,2294,3669,
1076,2707,370,2257,176,2319,494,2893,1324,3979,
2666,1481,424,3591,2790,2117,1572,1155,866,705,
672,767,990,1341,1820,2427,3162,4025,920,2039,
3286,565,2068,3699,1362,3249,1168,3311,1486,3885,
2316,875,3658,2473,1416,487,3782,3109,2564,2147,
1858,1697,1664,1759,1982,2333,2812,3419,58,921,
1912,3031,182,1557,3060,595,2354,145,2160,207,
2478,781,3308,1867,554,3465,2408,1479,678,5,
3556,3139,2850,2689,2656,2751,2974,3325,3804,315,
1050,1913,2904,4023,1174,2549,4052,1587,3346,1137,
3152,1199,3470,1773,204,2859,1546,361,3400,2471,
1670,997,452,35,3842,3681,3648,3743,3966,221,
700,1307,2042,2905,3896,919,2166,3541,948,2579,
242,2129,48,2191,366,2765,1196,3851,2538,1353,
296,3463,2662,1989,1444,1027,738,577,544,639,
862,1213,1692,2299,3034,3897,792,1911,3158,437,
1940,3571,1234,3121,1040,3183,1358,3757,2188,747,
3530,2345,1288,359,3654,2981,2436,2019,1730,1569,
1536,1631,1854,2205,2684,3291,4026,793,1784,2903,
54,1429,2932,467,2226,17,2032,79,2350,653,
3180,1739,426,3337,2280,1351,550,3973,3428,3011,
2722,2561,2528,2623,2846,3197,3676,187,922,1785,
2776,3895,1046,2421,3924,1459,3218,1009,3024,1071,
3342,1645,76,2731,1418,233,3272,2343,1542,869,
324,4003,3714,3553,3520,3615,3838,93,572,1179,
1914,2777,3768,791,2038,3413,820,2451,114,2001,
4016,2063,238,2637,1068,3723,2410,1225,168,3335,
2534,1861,1316,899,610,449,416,511,734,1085,
1564,2171,2906,3769,664,1783,3030,309,1812,3443,
1106,2993,912,3055,1230,3629,2060,619,3402,2217,
1160,231,3526,2853,2308,1891,1602,1441,1408,1503,
1726,2077,2556,3163,3898,665,1656,2775,4022,1301,
2804,339,2098,3985,1904,4047,2222,525,3052,1611,
298,3209,2152,1223,422,3845,3300,2883,2594,2433,
2400,2495,2718,3069,3548,59,794,1657,2648,3767,
918,2293,3796,1331,3090,881,2896,943,3214,1517,
4044,2603,1290,105,3144,2215,1414,741,196,3875,
3586,3425,3392,3487,3710,4061,444,1051,1786,2649,
3640,663,1910,3285,692,2323,4082,1873,3888,1935,
110,2509,940,3595,2282,1097,40,3207,2406,1733,
1188,771,482,321,288,383,606,957,1436,2043,
2778,3641,536,1655,2902,181,1684,3315,978,2865,
784,2927,1102,3501,1932,491,3274,2089,1032,103,
3398,2725,2180,1763,1474,1313,1280,1375,1598,1949,
2428,3035,3770,537,1528,2647,3894,1173,2676,211,
1970,3857,1776,3919,2094,397,2924,1483,170,3081,
2024,1095,294,3717,3172,2755,2466,2305,2272,2367,
2590,2941,3420,4027,666,1529,2520,3639,790,2165,
3668,1203,2962,753,2768,815,3086,1389,3916,2475,
1162,4073,3016,2087,1286,613,68,3747,3458,3297,
3264,3359,3582,3933,316,923,1658,2521,3512,535,
1782,3157,564,2195,3954,1745,3760,1807,4078,2381,
812,3467,2154,969,4008,3079,2278,1605,1060,643,
354,193,160,255,478,829,1308,1915,2650,3513,
408,1527,2774,53,1556,3187,850,2737,656,2799,
974,3373,1804,363,3146,1961,904,4071,3270,2597,
2052,1635,1346,1185,1152,1247,1470,1821,2300,2907,
3642,409,1400,2519,3766,1045,2548,83,1842,3729,
1648,3791,1966,269,2796,1355,42,2953,1896,967,
166,3589,3044,2627,2338,2177,2144,2239,2462,2813,
3292,3899,538,1401,2392,3511,662,2037,3540,1075,
2834,625,2640,687,2958,1261,3788,2347,1034,3945,
2888,1959,1158,485,4036,3619,3330,3169,3136,3231,
3454,3805,188,795,1530,2393,3384,407,1654,3029,
436,2067,3826,1617,3632,1679,3950,2253,684,3339,
2026,841,3880,2951,2150,1477,932,515,226,65,
32,127,350,701,1180,1787,2522,3385,280,1399,
2646,4021,1428,3059,722,2609,528,2671,846,3245,
1676,235,3018,1833,776,3943,3142,2469,1924,1507,
1218,1057,1024,1119,1342,1693,2172,2779,3514,281,
1272,2391,3638,917,2420,4051,1714,3601,1520,3663,
1838,141,2668,1227,4010,2825,1768,839,38,3461,
2916,2499,2210,2049,2016,2111,2334,2685,3164,3771,
410,1273,2264,3383,534,1909,3412,947,2706,497,
2512,559,2830,1133,3660,2219,906,3817,2760,1831,
1030,357,3908,3491,3202,3041,3008,3103,3326,3677,
60,667,1402,2265,3256,279,1526,2901,308,1939,
3698,1489,3504,1551,3822,2125,556,3211,1898,713,
3752,2823,2022,1349,804,387,98,4033,4000,4095,
222,573,1052,1659,2394,3257,152,1271,2518,3893,
1300,2931,594,2481,400,2543,718,3117,1548,107,
2890,1705,648,3815,3014,2341,1796,1379,1090,929,
896,991,1214,1565,2044,2651,3386,153,1144,2263,
3510,789,2292,3923,1586,3473,1392,3535,1710,13,
2540,1099,3882,2697,1640,711,4006,3333,2788,2371,
2082,1921,1888,1983,2206,2557,3036,3643,282,1145,
2136,3255,406,1781,3284,819,2578,369,2384,431,
2702,1005,3532,2091,778,3689,2632,1703,902,229,
3780,3363,3074,2913,2880,2975,3198,3549,4028,539,
1274,2137,3128,151,1398,2773,180,1811,3570,1361,
3376,1423,3694,1997,428,3083,1770,585,3624,2695,
1894,1221,676,259,4066,3905,3872,3967,94,445,
924,1531,2266,3129,24,1143,2390,3765,1172,2803,
466,2353,272,2415,590,2989,1420,4075,2762,1577,
520,3687,2886,2213,1668,1251,962,801,768,863,
1086,1437,1916,2523,3258,25,1016,2135,3382,661,
2164,3795,1458,3345,1264,3407,1582,3981,2412,971,
3754,2569,1512,583,3878,3205,2660,2243,1954,1793,
1760,1855,2078,2429,2908,3515,154,1017,2008,3127,
278,1653,3156,691,2450,241,2256,303,2574,877,
3404,1963,650,3561,2504,1575,774,101,3652,3235,
2946,2785,2752,2847,3070,3421,3900,411,1146,2009,
3000,23,1270,2645,52,1683,3442,1233,3248,1295,
3566,1869,300,2955,1642,457,3496,2567,1766,1093,
548,131,3938,3777,3744,3839,4062,317,796,1403,
2138,3001,3992,1015,2262,3637,1044,2675,338,2225,
144,2287,462,2861,1292,3947,2634,1449,392,3559,
2758,2085,1540,1123,834,673,640,735,958,1309,
1788,2395,3130,3993,888,2007,3254,533,2036,3667,
1330,3217,1136,3279,1454,3853,2284,843,3626,2441,
1384,455,3750,3077,2532,2115,1826,1665,1632,1727,
1950,2301,2780,3387,26,889,1880,2999,150,1525,
3028,563,2322,113,2128,175,2446,749,3276,1835,
522,3433,2376,1447,646,4069,3524,3107,2818,2657,
2624,2719,2942,3293,3772,283,1018,1881,2872,3991,
1142,2517,4020,1555,3314,1105,3120,1167,3438,1741,
172,2827,1514,329,3368,2439,1638,965,420,3,
3810,3649,3616,3711,3934,189,668,1275,2010,2873,
3864,887,2134,3509,916,2547,210,2097,16,2159,
334,2733,1164,3819,2506,1321,264,3431,2630,1957,
1412,995,706,545,512,607,830,1181,1660,2267,
3002,3865,760,1879,3126,405,1908,3539,1202,3089,
1008,3151,1326,3725,2156,715,3498,2313,1256,327,
3622,2949,2404,1987,1698,1537,1504,1599,1822,2173,
2652,3259,3994,761,1752,2871,22,1397,2900,435,
2194,4081,2000,47,2318,621,3148,1707,394,3305,
2248,1319,518,3941,3396,2979,2690,2529,2496,2591,
2814,3165,3644,155,890,1753,2744,3863,1014,2389,
3892,1427,3186,977,2992,1039,3310,1613,44,2699,
1386,201,3240,2311,1510,837,292,3971,3682,3521,
3488,3583,3806,61,540,1147,1882,2745,3736,759,
2006,3381,788,2419,82,1969,3984,2031,206,2605,
1036,3691,2378,1193,136,3303,2502,1829,1284,867,
578,417,384,479,702,1053,1532,2139,2874,3737,
632,1751,2998,277,1780,3411,1074,2961,880,3023,
1198,3597,2028,587,3370,2185,1128,199,3494,2821,
2276,1859,1570,1409,1376,1471,1694,2045,2524,3131,
3866,633,1624,2743,3990,1269,2772,307,2066,3953,
1872,4015,2190,493,3020,1579,266,3177,2120,1191,
390,3813,3268,2851,2562,2401,2368,2463,2686,3037,
3516,27,762,1625,2616,3735,886,2261,3764,1299,
3058,849,2864,911,3182,1485,4012,2571,1258,73,
3112,2183,1382,709,164,3843,3554,3393,3360,3455,
3678,4029,412,1019,1754,2617,3608,631,1878,3253,
660,2291,4050,1841,3856,1903,78,2477,908,3563,
2250,1065,8,3175,2374,1701,1156,739,450,289,
256,351,574,925,1404,2011,2746,3609,504,1623,
2870,149,1652,3283,946,2833,752,2895,1070,3469,
1900,459,3242,2057,1000,71,3366,2693,2148,1731,
1442,1281,1248,1343,1566,1917,2396,3003,3738,505,
1496,2615,3862,1141,2644,179,1938,3825,1744,3887,
2062,365,2892,1451,138,3049,1992,1063,262,3685,
3140,2723,2434,2273,2240,2335,2558,2909,3388,3995,
634,1497,2488,3607,758,2133,3636,1171,2930,721,
2736,783,3054,1357,3884,2443,1130,4041,2984,2055,
1254,581,36,3715,3426,3265,3232,3327,3550,3901,
284,891,1626,2489,3480,503,1750,3125,532,2163,
3922,1713,3728,1775,4046,2349,780,3435,2122,937,
3976,3047,2246,1573,1028,611,322,161,128,223,
446,797,1276,1883,2618,3481,376,1495,2742,21,
1524,3155,818,2705,624,2767,942,3341,1772,331,
3114,1929,872,4039,3238,2565,2020,1603,1314,1153,
1120,1215,1438,1789,2268,2875,3610,377,1368,2487,
3734,1013,2516,51,1810,3697,1616,3759,1934,237,
2764,1323,10,2921,1864,935,134,3557,3012,2595,
2306,2145,2112,2207,2430,2781,3260,3867,506,1369,
2360,3479,630,2005,3508,1043,2802,593,2608,655,
2926,1229,3756,2315,1002,3913,2856,1927,1126,453,
4004,3587,3298,3137,3104,3199,3422,3773,156,763,
1498,2361,3352,375,1622,2997,404,2035,3794,1585,
3600,1647,3918,2221,652,3307,1994,809,3848,2919,
2118,1445,900,483,194,33};





double gaussian(double variance)
{
	// static becuase we don't want to have it initialized each time we go in
	double returnvalue=0;
	double k;
	
	k = sqrt(variance/2.0);
	
	// add 24 uniform RV to obtain a simulation of normality
    int x;
	for (x=0;x<24;x++)
		returnvalue += doublerandom();
	
	return k*(returnvalue-0.5*24);

}




//////////////////////////////////////////////////////////////////////
// block interleave
// L_TOTAL must = x^2,otherwise,who knows?
//////////////////////////////////////////////////////////////////////
void init_Block_interleave_table()
{
	INT i,j;
	INT temp;

	temp = (INT)sqrt(L_TOTAL);
	for (i=0;i<temp;i++)
		for (j=0;j<temp;j++)
			m_Inter_table[i*temp+j] = j*temp+i;

	
}

//////////////////////////////////////////////////////////////////////
// RSC endcoder
// mesg -- {0,1}
// parity -- {0,1}
// force==1,terminated --- for outer encoder
//////////////////////////////////////////////////////////////////////
void RSC_Encode(BYTE *mesg, BYTE *parity, unsigned int size, bool force)
{
	BYTE state,uk;
	unsigned x;
	
	state=0;
	for (x=0;x<size;x++)
	{
		// force the encoder to zero state at the end
		if (x>=size-M && force)
		{
			mesg[x] = TailBit[state];
		}
		
		// can't assume the bool type has an intrinsic value of 0 or 1
		// may differ from platform to platform
		uk = mesg[x] ? 1 : 0;
		
		// calculate output due to new mesg bit
		parity[x] = EnNextOut[uk][state];
		// calculate the new state
		state = NextState[uk][state];
	}
}


//////////////////////////////////////////////////////////////////////
// Turbo encoder
// msg -- {0,1}
// stream -- {0,1}
// puncture -- true to get 1/2 rate,NOT tested yet
//////////////////////////////////////////////////////////////////////
void encode(BYTE *msg, BYTE *stream, bool puncture)
{
	INT i;
	BYTE imsg[L_TOTAL];
	BYTE chkBuffer[2][L_TOTAL];
	// first encoder
	RSC_Encode(msg,chkBuffer[0],L_TOTAL,true);
	// interleave
	for (i=0;i<L_TOTAL;i++)
		imsg[i]=msg[m_Inter_table[i]];
	// second encoder
	RSC_Encode(imsg,chkBuffer[1],L_TOTAL,false);
	// punture
	for (i=0;i<L_TOTAL;i++)
	{
		if(puncture){
			stream[i*2]=msg[i];
			stream[i*2+1]=chkBuffer[i%2][i];
		}else{
			stream[i*3]=msg[i];
			stream[i*3+1]=chkBuffer[0][i];
			stream[i*3+2]=chkBuffer[1][i];
		}	
	}
}


__global__ void interLeave(double * src, double * des , unsigned int * interLeaveTable ){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    des[tid] = src[interLeaveTable[tid]];
}

__global__ void deInterLeave(double * src, double * des , unsigned int * interLeaveTable ){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    des[interLeaveTable[tid]] = src[tid];
}

__global__ void gammaAlpha(double * msg ,double * parity, double * L_a, double (*gamma)[8][8], BYTE (*lastState)[8],char (*lastOut)[8] ){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;

    unsigned int s0,s2;
    for (s0=0;s0<NSTATE;s0++) {
		for (s2=0;s2<NSTATE;s2++)
			gamma[tid][s0][s2]=-INIFINITY;
		gamma[tid][s0][lastState[0][s0]]=-msg[tid]+parity[tid]*lastOut[0][s0]-log(1+exp(L_a[tid]));
		gamma[tid][s0][lastState[1][s0]]=msg[tid]+parity[tid]*lastOut[1][s0]+L_a[tid]-log(1+exp(L_a[tid]));
		//gamma[tid][s0][lastState[0][s0]]=0.5;
		//gamma[tid][s0][lastState[1][s0]]=-0.5;
    }
}

__global__ void gammaBeta(double * msg ,double * parity, double * L_a, double (*gamma)[8][8], BYTE (*nextState)[8], char (*nextOut)[8]){
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;

    unsigned int s0,s2;
    for (s0=0;s0<NSTATE;s0++) {
		for (s2=0;s2<NSTATE;s2++)
			gamma[tid][s0][s2]=-INIFINITY;
		gamma[tid][s0][nextState[0][s0]]=-msg[tid]+parity[tid]*nextOut[0][s0]-log(1+exp(L_a[tid]));
		gamma[tid][s0][nextState[1][s0]]=msg[tid]+parity[tid]*nextOut[1][s0]+L_a[tid]-log(1+exp(L_a[tid]));
		//gamma[tid][s0][nextState[0][s0]]=0.5;
		//gamma[tid][s0][nextState[1][s0]]=-0.5;
    }
}

__global__ void Alpha(double (*Alpha)[8], double (*gamma)[8][8], double *maxBranch) {
	const int tid = blockIdx.x*blockDim.x + threadIdx.x;

	UINT k, s1, s2;
	double sum;

	if (tid == 0) {
		Alpha[0][0] = 0.0;
		for (s1=1;s1<NSTATE;s1++)
			Alpha[0][s1]=-INIFINITY;
	}
	else {
		for (s1=0;s1<NSTATE;s1++)
			Alpha[tid*(L_TOTAL/AlphaBetaTHREAD_NUM)][s1]=0;
	}

	//for (k=1; k<=L_TOTAL; k++) {
	for (k=tid*L_TOTAL/AlphaBetaTHREAD_NUM+1; k<(tid*L_TOTAL/AlphaBetaTHREAD_NUM+L_TOTAL/AlphaBetaTHREAD_NUM); k++) {
        for (s2=0;s2<NSTATE;s2++){
            sum = 0.0;
            for (s1=0;s1<NSTATE;s1++) {
                sum+=exp(gamma[k-1][s2][s1]+Alpha[k-1][s1]);
			}
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                Alpha[k][s2]=-INIFINITY;
            else
                Alpha[k][s2]=log(sum);
        }

		// normalization,prevent overflow
		maxBranch[k]=Alpha[k][0];
		for (s2=1;s2<NSTATE;s2++)
			if (Alpha[k][s2]>maxBranch[k])
				maxBranch[k]=Alpha[k][s2];

		for (s2=0;s2<NSTATE;s2++)
			Alpha[k][s2]=Alpha[k][s2]-maxBranch[k];
	}

}

__global__ void Beta(double (*Beta)[8], double (*gamma)[8][8], bool index, double* maxBranch) {
	const int tid = blockIdx.x*blockDim.x + threadIdx.x;

	UINT k, s1, s2;
	double sum;

	if (tid == (AlphaBetaTHREAD_NUM-1)) {
		if (index){// true -- terminated,false -- open
        Beta[L_TOTAL][0]=0.0;
        for (s2=1;s2<NSTATE;s2++)
            Beta[L_TOTAL][s2]=-INIFINITY;
		}
		else 
			for (s2=0;s2<NSTATE;s2++)
				Beta[L_TOTAL][s2]=0;
	}
	else {
		for (s2=0; s2<NSTATE; s2++)
			Beta[(tid+1)*L_TOTAL/AlphaBetaTHREAD_NUM][s2]=0;
	}

    for (k=(tid+1)*L_TOTAL/AlphaBetaTHREAD_NUM-1;k>(tid*L_TOTAL/AlphaBetaTHREAD_NUM);k--) {
   // for (k=L_TOTAL-1;k>0;k--) {

        for (s1=0;s1<NSTATE;s1++) {
            sum = 0.0;
            for (s2=0;s2<NSTATE;s2++) 
                sum += exp(gamma[k][s1][s2] + Beta[k+1][s2]);
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                Beta[k][s1] = -INIFINITY;
            else 
                Beta[k][s1] = log(sum);
        }

		// normalization,prevent overflow
		for (s2=0;s2<NSTATE;s2++)
			Beta[k][s2]=Beta[k][s2]-maxBranch[k];
	}
}

void computeAlpha(double (*AlphaHost)[8], double (*gamma)[8][8], double *maxBranch) {
    // initialize Alpha & Beta
    AlphaHost[0][0]=0;
	UINT s1,k,s2;
	double sum;
    for (s1=1;s1<NSTATE;s1++)
        AlphaHost[0][s1]=-INIFINITY;

    for (k=1;k<=L_TOTAL;k++){

        for (s2=0;s2<NSTATE;s2++){
            sum = 0;
            for (s1=0;s1<NSTATE;s1++) {
                sum+=exp(gamma[k-1][s2][s1]+AlphaHost[k-1][s1]);
			}
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                AlphaHost[k][s2]=-INIFINITY;
            else
                AlphaHost[k][s2]=log(sum);
        }

		// normalization,prevent overflow
		maxBranch[k]=AlphaHost[k][0];
		for (s2=1;s2<NSTATE;s2++)
			if (AlphaHost[k][s2]>maxBranch[k])
				maxBranch[k]=AlphaHost[k][s2];

		for (s2=0;s2<NSTATE;s2++)
			AlphaHost[k][s2]=AlphaHost[k][s2]-maxBranch[k];
    }
}

void computeBeta(double (*BetaHost)[8], double (*gamma)[8][8], bool index, double * maxBranch){
    // initialize Beta
	UINT s1,k,s2;
	double sum;
    if (index){// true -- terminated,false -- open
        BetaHost[L_TOTAL][0]=0;
        for (s2=1;s2<NSTATE;s2++)
            BetaHost[L_TOTAL][s2]=-INIFINITY;
    }
    else 
        for (s2=0;s2<NSTATE;s2++)
            BetaHost[L_TOTAL][s2]=0;

    for (k=L_TOTAL-1;k>0;k--) {

        for (s1=0;s1<NSTATE;s1++) {
            sum = 0.0;
            for (s2=0;s2<NSTATE;s2++) 
                sum += exp(gamma[k][s1][s2] + BetaHost[k+1][s2]);
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                BetaHost[k][s1] = -INIFINITY;
            else 
                BetaHost[k][s1] = log(sum);
        }

		// normalization,prevent overflow
		for (s2=0;s2<NSTATE;s2++)
			BetaHost[k][s2]=BetaHost[k][s2]-maxBranch[k];
    }
}

__global__ void normalizationAlphaAndBeta(double (*Alpha)[8], double (*Beta)[8]) {
    unsigned int tid = threadIdx.x+1; 
    double max_branch;
    max_branch = Alpha[tid][0];
	UINT s2;
    for (s2=1;s2<NSTATE;s2++)
        if (Alpha[tid][s2]>max_branch)
            max_branch = Alpha[tid][s2];

    for (s2=0;s2<NSTATE;s2++) {
        Alpha[tid][s2] = Alpha[tid][s2] - max_branch;

        if (tid != L_TOTAL) 
            Beta[tid][s2] = Beta[tid][s2] - max_branch;
    }

}

__global__ void LLRS(double * msg, double * parity, double * L_a, double (*Alpha)[8], double (*Beta)[8], double * L_all,BYTE (*lastState)[8], char (*lastOut)[8]) {
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    UINT s2;
	double sum0 = 0.0, sum1 = 0.0;
    for (s2=0;s2<NSTATE;s2++) {
        //gamma[LastState[0][s2]]=-msg[tid]+parity[tid]*LastOut[0][s2]-log(1+exp(L_a[tid]));
        //gamma[LastState[1][s2]]=msg[tid]+parity[tid]*LastOut[1][s2]+L_a[tid]-log(1+exp(L_a[tid]));
        //sum0+=exp(gamma[LastState[0][s2]]+Alpha[tid][LastState[0][s2]]+Beta[tid+1][s2]);
        //sum1+=exp(gamma[LastState[1][s2]]+Alpha[tid][LastState[1][s2]]+Beta[tid+1][s2]);
        double gamma0=-msg[tid]+parity[tid]*lastOut[0][s2]-log(1+exp(L_a[tid]));
        double gamma1=msg[tid]+parity[tid]*lastOut[1][s2]+L_a[tid]-log(1+exp(L_a[tid]));
        sum0+=exp(gamma0+Alpha[tid][lastState[0][s2]]+Beta[tid+1][s2]);
        sum1+=exp(gamma1+Alpha[tid][lastState[1][s2]]+Beta[tid+1][s2]);
    }
    //L_all[tid]=log(sum1)-log(sum0);
    L_all[tid]=log(sum1)-log(sum0);
}

__global__ void extrinsicInformation(double * L_all, double * msg, double * L_a, double * L_e) {
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    L_e[tid] = L_all[tid] - 2*msg[tid] - L_a[tid];
}

__global__ void demultiplex(double * stream, double * msg, double * parity0, double * parity1) {
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    //if (puncture){// punctured rate=1/2
    //    msg[tid]=stream[2*tid];
    //    parity[tid%2][tid]=stream[tid*2+1];
    //}
    //else {// unpunctured rate=1/3
    //    msg[tid]=stream[3*tid];
    //    parity0[tid]=stream[3*tid+1];
    //    parity1[tid]=stream[3*tid+2];
    //}
        msg[tid]=stream[3*tid];
        parity0[tid]=stream[3*tid+1];
        parity1[tid]=stream[3*tid+2];
}

__global__ void initializeExtrinsicInformation(double * L_e) {
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    L_e[tid] = 0;
    
}

__global__ void exestimateInformationBits(double * L_all, BYTE * msghat, UINT * m_Inter_table) {
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(L_all[tid]>0)
        msghat[m_Inter_table[tid]]=1;
    else
        msghat[m_Inter_table[tid]]=0;
}

void countErrors(BYTE *m, BYTE * mhat, UINT * bitsError, UINT * frameError, UINT iter) {

	bool f_err = false;
	for (int i=0; i<(L_TOTAL-M);i++) {
		if (m[i] != mhat[i]) {
			bitsError[iter] = bitsError[iter]+1;
			f_err = true;
		}
	}

	if (f_err) 
		frameError[iter] = frameError[iter]+1;
}


int main(int argc, char* argv[])
{
    initRandom(seed);

	BYTE * m;
	BYTE * x;
	double * y;
	BYTE * mhat;

	int frame;
	UINT bits_all,bits_err[MAXITER],frame_err[MAXITER];
	double Ber,Fer;
	double Eb_No_dB,No;
	//bool f_err;
	//FILE * fp;
	int i;

	m = new BYTE[L_TOTAL];
	x = new BYTE[L_ALL];
	y = new double[L_ALL];
	mhat = new BYTE[L_TOTAL];

	//init_Block_interleave_table();	// block interleave


    
    findCudaDevice(argc, (const char **)argv);

	BYTE (*LastStateDevice)[8];
	BYTE (*NextStateDevice)[8];
	char (*LastOutDevice)[8];
	char (*NextOutDevice)[8];
	double * yDevice;
	double * msgDevice;
	double * imsgDevice;
	BYTE * mhatDevice;
	double * parity0Device;
	double * parity1Device;
	UINT * tableDevice;
	double * L_eDevice;
	double * L_aDevice;
	double * L_allDevice;
	double (*gammaAlphaDevice)[8][8];
	double (*gammaBetaDevice)[8][8];
	double (*AlphaDevice)[8];
	double (*BetaDevice)[8];
	double *maxBranchDevice;

    hipMalloc((void **)&LastStateDevice, 2*8*sizeof(BYTE));
    hipMalloc((void **)&NextStateDevice, 2*8*sizeof(BYTE));
    hipMalloc((void **)&LastOutDevice, 2*8*sizeof(char));
    hipMalloc((void **)&NextOutDevice, 2*8*sizeof(char));

    hipMalloc((void **)&yDevice, L_ALL*sizeof(double));
    hipMalloc((void **)&msgDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&imsgDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&mhatDevice, L_TOTAL*sizeof(BYTE));
    hipMalloc((void **)&parity0Device, L_TOTAL*sizeof(double));
    hipMalloc((void **)&parity1Device, L_TOTAL*sizeof(double));
    hipMalloc((void **)&tableDevice, L_TOTAL*sizeof(unsigned int));
    hipMalloc((void **)&L_eDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&L_aDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&L_allDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&gammaAlphaDevice, L_TOTAL*sizeof(double)*8*8);
    hipMalloc((void **)&gammaBetaDevice, L_TOTAL*sizeof(double)*8*8);
    hipMalloc((void **)&AlphaDevice, (L_TOTAL+1)*sizeof(double)*8);
    hipMalloc((void **)&BetaDevice, (L_TOTAL+1)*sizeof(double)*8);
    hipMalloc((void **)&maxBranchDevice, (L_TOTAL+1)*sizeof(double));

	//For Debug
	//double L_aHost[L_TOTAL];
	//double L_aHost1[L_TOTAL];
	//double L_allHost[L_TOTAL];

    double gammaAlphaHost[L_TOTAL][8][8];
    double gammaBetaHost[L_TOTAL][8][8];
    double AlphaHost[L_TOTAL+1][8];
    double BetaHost[L_TOTAL+1][8];

	double max_branch[L_TOTAL+1];

    hipMemcpy(LastStateDevice,LastState,sizeof(BYTE)*2*8, hipMemcpyHostToDevice);
    hipMemcpy(NextStateDevice,NextState,sizeof(BYTE)*2*8, hipMemcpyHostToDevice);
    hipMemcpy(LastOutDevice,LastOut,sizeof(char)*2*8, hipMemcpyHostToDevice);
    hipMemcpy(NextOutDevice,NextOut,sizeof(char)*2*8, hipMemcpyHostToDevice);

    hipMemcpy(tableDevice,m_Inter_table,sizeof(unsigned int)*L_TOTAL, hipMemcpyHostToDevice);

	for (Eb_No_dB= 0.0;Eb_No_dB<3.0;Eb_No_dB+=0.5){

	//Eb_No_dB = 0.0;
		No = 1/pow(10.0,Eb_No_dB/10.0);
		bits_all = 0;
		for (i =0; i<MAXITER;i++) {
			bits_err[i]=0;
			frame_err[i]=0;
		}

		for (frame = 0; frame<FRAME_NUM; frame++, bits_all += (L_TOTAL-M)) {

			// Generate random information bits
			for (i=0;i<L_TOTAL;i++)
				if (boolrandom())
					m[i]=1;
				else
					m[i]=0;
			// encoder
			encode(m,x,false);
			// add noise
			for (i=0;i<L_ALL;i++)
				if (x[i])
					y[i]=1.0+gaussian(No/2);
				else
					y[i]=-1.0+gaussian(No/2);

			hipMemcpy(yDevice,y,sizeof(double)*L_ALL, hipMemcpyHostToDevice);

			demultiplex<<<BLOCK_NUM,THREAD_NUM>>>(yDevice, msgDevice, parity0Device, parity1Device); 
			interLeave<<<BLOCK_NUM,THREAD_NUM>>>(msgDevice, imsgDevice, tableDevice);
			initializeExtrinsicInformation<<<BLOCK_NUM,THREAD_NUM>>>(L_eDevice);

			for (int iter = 0; iter<MAXITER; iter++) {
				
				deInterLeave<<<BLOCK_NUM,THREAD_NUM>>>(L_eDevice, L_aDevice, tableDevice);

				gammaAlpha<<<BLOCK_NUM,THREAD_NUM>>>(msgDevice , parity0Device,  L_aDevice,  gammaAlphaDevice,LastStateDevice, LastOutDevice);
				gammaBeta<<<BLOCK_NUM,THREAD_NUM>>>(msgDevice , parity0Device,  L_aDevice,  gammaBetaDevice, NextStateDevice, NextOutDevice);
				Alpha<<<1,AlphaBetaTHREAD_NUM>>>(AlphaDevice, gammaAlphaDevice, maxBranchDevice);
				Beta<<<1,AlphaBetaTHREAD_NUM>>>(BetaDevice, gammaBetaDevice,true, maxBranchDevice);
				//hipMemcpy(AlphaHost, AlphaDevice, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyDeviceToHost);
				//hipMemcpy(gammaAlphaHost, gammaAlphaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);
				//hipMemcpy(gammaBetaHost, gammaBetaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);

				//computeAlpha(AlphaHost, gammaAlphaHost, max_branch);
				//computeBeta(BetaHost, gammaBetaHost, true,max_branch);
				//hipMemcpy(AlphaDevice, AlphaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				//hipMemcpy(BetaDevice, BetaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				//normalizationAlphaAndBeta<<<BLOCK_NUM,THREAD_NUM>>>(AlphaDevice, BetaDevice);
				//hipMemcpy(AlphaHost, AlphaDevice, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyDeviceToHost);

				LLRS<<<BLOCK_NUM,THREAD_NUM>>>(msgDevice, parity0Device, L_aDevice, AlphaDevice, BetaDevice, L_allDevice,LastStateDevice, LastOutDevice);

				extrinsicInformation<<<BLOCK_NUM,THREAD_NUM>>>(L_allDevice, msgDevice, L_aDevice, L_eDevice);
				//if (iter >= 3) {
				///debug
				//exestimateInformationBits<<<BLOCK_NUM,THREAD_NUM>>>(L_allDevice, mhatDevice, tableDevice); 

				//hipMemcpy(mhat, mhatDevice, sizeof(BYTE)*L_TOTAL, hipMemcpyDeviceToHost);
				//countErrors(m, mhat, bits_err, frame_err, iter);
				//debug
				//hipMemcpy(L_aHost1, L_aDevice, sizeof(double)*L_TOTAL, hipMemcpyDeviceToHost);
				//}

				interLeave<<<BLOCK_NUM,THREAD_NUM>>>(L_eDevice, L_aDevice, tableDevice);

				gammaAlpha<<<BLOCK_NUM,THREAD_NUM>>>(imsgDevice , parity1Device,  L_aDevice,  gammaAlphaDevice, LastStateDevice, LastOutDevice);
				gammaBeta<<<BLOCK_NUM,THREAD_NUM>>>(imsgDevice , parity1Device,  L_aDevice,  gammaBetaDevice, NextStateDevice, NextOutDevice);
				Alpha<<<1,AlphaBetaTHREAD_NUM>>>(AlphaDevice, gammaAlphaDevice, maxBranchDevice);
				Beta<<<1,AlphaBetaTHREAD_NUM>>>(BetaDevice, gammaBetaDevice,false, maxBranchDevice);
				//hipMemcpy(AlphaHost, AlphaDevice, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyDeviceToHost);
				//hipMemcpy(gammaAlphaHost, gammaAlphaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);
				//hipMemcpy(gammaBetaHost, gammaBetaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);

				//computeAlpha(AlphaHost, gammaAlphaHost, max_branch);
				//Beta(BetaHost, gammaBetaHost, false, max_branch);
				//hipMemcpy(AlphaDevice, AlphaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				//hipMemcpy(BetaDevice, BetaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				//normalizationAlphaAndBeta<<<BLOCK_NUM,THREAD_NUM>>>(AlphaDevice, BetaDevice);
				//hipMemcpy(AlphaHost, AlphaDevice, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyDeviceToHost);

				LLRS<<<BLOCK_NUM,THREAD_NUM>>>(imsgDevice, parity1Device, L_aDevice, AlphaDevice, BetaDevice, L_allDevice, LastStateDevice,LastOutDevice);

				extrinsicInformation<<<BLOCK_NUM,THREAD_NUM>>>(L_allDevice, imsgDevice, L_aDevice, L_eDevice);

				exestimateInformationBits<<<BLOCK_NUM,THREAD_NUM>>>(L_allDevice, mhatDevice, tableDevice); 

				hipMemcpy(mhat, mhatDevice, sizeof(BYTE)*L_TOTAL, hipMemcpyDeviceToHost);
				countErrors(m, mhat, bits_err, frame_err, iter);

				//debug
				//hipMemcpy(L_aHost, L_aDevice, sizeof(double)*L_TOTAL, hipMemcpyDeviceToHost);
			}
			// estimate information bits
			//exestimateInformationBits<<<BLOCK_NUM,THREAD_NUM>>>(L_allDevice, mhatDevice, tableDevice); 

			//hipMemcpy(mhat, mhatDevice, sizeof(BYTE)*L_TOTAL, hipMemcpyDeviceToHost);
			// count errors
			//UINT bits_err = 0;
			//for (i=0;i<L_TOTAL-M;i++) {
			//	if (mhat[i]!=m[i]) {
			//		bits_err++;
			//	}
			//}
			//cout<<"bits_err: "<<bits_err<<endl;
		}

		printf("-------------------------\n");
		printf("Eb/No=%fdB:\n",Eb_No_dB);
		printf("-------------------------\n");
		//fprintf(fp,"-------------------------\n");
		//fprintf(fp,"Eb/No=%fdB:\n",Eb_No_dB);
		//fprintf(fp,"-------------------------\n");

		for (i=0;i<MAXITER;i++) {
			Ber=(double)bits_err[i]/(double)bits_all;
			Fer=(double)frame_err[i]/(double)FRAME_NUM;
			printf("Iteration:%d\n",i+1);
			printf("---Ber=%f\n---Fer=%f\n",Ber,Fer);
			//fprintf(fp,"Iteration:%d\n",i);
			//fprintf(fp,"---Ber=%f\n---Fer=%f\n",Ber,Fer);
		}
	}

	delete m;
	delete x;
	delete y;
	delete mhat;

	hipFree(LastStateDevice);
	hipFree(NextStateDevice);
	hipFree(LastOutDevice);
	hipFree(NextOutDevice);
	hipFree(yDevice);
	hipFree(msgDevice);
	hipFree(imsgDevice);
	hipFree(mhatDevice);
	hipFree(parity0Device);
	hipFree(parity1Device);
	hipFree(tableDevice);
	hipFree(L_eDevice);
	hipFree(L_aDevice);
	hipFree(L_allDevice);
	hipFree(gammaAlphaDevice);
	hipFree(gammaBetaDevice);
	hipFree(AlphaDevice);
	hipFree(BetaDevice);

	return 0;
}
