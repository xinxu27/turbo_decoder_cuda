#include "hip/hip_runtime.h"
// TurboDecoder : Defines the entry point for the console application.
#include "hip/hip_runtime_api.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <limits.h>
#include <iostream>
using namespace std;


#define IM1 2147483563
#define IM2 2147483399
#define AM (1.0/IM1)
#define IMM1 (IM1-1)
#define IA1 40014
#define IA2 40692
#define IQ1 53668
#define IQ2 52774
#define IR1 12211
#define IR2 3791
#define NTAB 32
#define NDIV (1+IMM1/NTAB)
#define EPS 1.7E-308 //MINDOUBLE
#define RNMX (1.0-EPS)
#define INIFINITY  1E+10

#define MIN 1E-300

//typedef enum __bool { false = 0, true = 1, } bool;

long idum2;
long idum;
long iy;
long iv[NTAB];	
unsigned memory;

/*
Long period (? 2 \Theta 10 18 ) random number generator of L'Ecuyer with Bays­Durham shuffle
and added safeguards. Returns a uniform random deviate between 0.0 and 1.0 (exclusive of
the endpoint values). Call with idum a negative integer to initialize; thereafter, do not alter
idum between successive deviates in a sequence. RNMX should approximate the largest floating
value that is less than 1.
--
*/


double ran2()
{
	int j;
	long k;
	double temp;
	
	
	k=(idum)/IQ1;
	idum=IA1*(idum-k*IQ1)-k*IR1;  // Compute idum=(IA1*idum) % IM1 without overflows by Schrage's method.
	if (idum < 0)
		idum += IM1;
	k=idum2/IQ2;
	idum2=IA2*(idum2-k*IQ2)-k*IR2;	// Compute idum2=(IA2*idum) % IM2 likewise.
	if (idum2 < 0)
		idum2 += IM2;
	j = iy/NDIV;
	iy=iv[j]-idum2;
	//	iy=iv[j]-idum2; 		// Here idum is shuffled, idum and idum2 are combined to generate output.
	iv[j] = idum;
	if (iy < 1)
		iy += IMM1;
	if ((temp=AM*iy) > RNMX)
		return RNMX; 		// Because users don't expect endpoint values.
	else
		return temp;
}


void initRandom(long seed)
{
	idum2=123456789;
	idum=0;
	iy=0;
	
	if (seed != 0)
		idum = seed;
	else
		idum = 1;
	
	
    int j;
	for (j=NTAB+7;j>=0;j--) // Load the shuffle table (after 8 warm­ups).
	{
		long k=(idum)/IQ1;
		
		idum=IA1*(idum-k*IQ1)-k*IR1;
		if (idum < 0)
			idum += IM1;
		if (j < NTAB)
			iv[j] = idum;
	}
	iy=iv[0];
}


double doublerandom()
{
	double t = ran2();
	return t;
}

long longrandom(long range)
{
	double t;
	
	t = doublerandom();
	return((long)(t*(double)range));
}


bool boolrandom()
{
	double t=doublerandom();
	
	if (t>0.5)
		return true;
	else
		return false;
}
//time_t t;	
//time(&t);	
//init((long)t);
long seed = 1234421;




#define L_TOTAL 6144// if u want to use block interleave,L_TOTAL must = x^2
#define M	3	// register length,=tail length
#define NSTATE	8	// = M^2
#define L_ALL 3*L_TOTAL	// coded frame length
//#define DELTA 30	// SOVA window size. Make decision after 'delta' delay. Decide bit k when received bits
					// for bit (k+delta) are processed. Trace back from (k+delta) to k. 
// Rate 1/3 Turbo code
// The generator polynomials used are:
//	g0=[1 1 1]
//  g1=[1 0 1]
//	RSC encoder structure:
//
//			 +------------------------------------------>c1
//           |          g0(x)    
//           x_.--o-------------(+)<----------+
//           |    |             /|\           |
//			 |   \|/      ---    |     ---    |
// data --_.-o-->(+)--o->| D0|---o--->| D1|---o
//                    |   ---          ---    |
//                    |                       |
//                    +-------->(+)<----------+
//                      g1(x)    |
//								 +---------------------->c2
//
typedef unsigned char BYTE;
typedef int INT;
typedef unsigned int UINT;
typedef int BOOL;

// NextOut[bk][current state]
static const char EnNextOut[2][NSTATE] = // check bit based on current and input bit
{	0,0,1,1,1,1,0,0,
	0,0,1,1,1,1,0,0
};
static const char NextOut[2][NSTATE] = // check bit based on current and input bit
{	-1,-1,1,1,1,1,-1,-1,
	-1,-1,1,1,1,1,-1,-1
};
// NextState[bk][current state]
static const BYTE NextState[2][NSTATE] = // next state based on current and input bit
{	0,4,5,1,2,6,7,3,
	4,0,1,5,6,2,3,7
};
// LastOut[bk][current state]
static const char LastOut[2][NSTATE] =	// trellis last check bit
{	-1,1,1,-1,-1,1,1,-1,
	-1,1,1,-1,-1,1,1,-1
};
// LastState[bk][current state]
static const BYTE LastState[2][NSTATE] =	// last state lead to current state by input bk
{	0,3,4,7,1,2,5,6,
	1,2,5,6,0,3,4,7
};
// TailBit[current state]
static const char TailBit[NSTATE] = // tail info bits when trellis is terminating
{	0,1,1,0,0,1,1,0
};


#define MAXITER 5
#define	FRAME_NUM 10

UINT m_Inter_table[L_TOTAL];





double gaussian(double variance)
{
	// static becuase we don't want to have it initialized each time we go in
	double returnvalue=0;
	double k;
	
	k = sqrt(variance/2.0);
	
	// add 24 uniform RV to obtain a simulation of normality
    int x;
	for (x=0;x<24;x++)
		returnvalue += doublerandom();
	
	return k*(returnvalue-0.5*24);

}




//////////////////////////////////////////////////////////////////////
// block interleave
// L_TOTAL must = x^2,otherwise,who knows?
//////////////////////////////////////////////////////////////////////
void init_Block_interleave_table()
{
	INT i,j;
	INT temp;

	temp = (INT)sqrt(L_TOTAL);
	for (i=0;i<temp;i++)
		for (j=0;j<temp;j++)
			m_Inter_table[i*temp+j] = j*temp+i;

	
}

//////////////////////////////////////////////////////////////////////
// RSC endcoder
// mesg -- {0,1}
// parity -- {0,1}
// force==1,terminated --- for outer encoder
//////////////////////////////////////////////////////////////////////
void RSC_Encode(BYTE *mesg, BYTE *parity, unsigned int size, bool force)
{
	BYTE state,uk;
	unsigned x;
	
	state=0;
	for (x=0;x<size;x++)
	{
		// force the encoder to zero state at the end
		if (x>=size-M && force)
		{
			mesg[x] = TailBit[state];
		}
		
		// can't assume the bool type has an intrinsic value of 0 or 1
		// may differ from platform to platform
		uk = mesg[x] ? 1 : 0;
		
		// calculate output due to new mesg bit
		parity[x] = EnNextOut[uk][state];
		// calculate the new state
		state = NextState[uk][state];
	}
}


//////////////////////////////////////////////////////////////////////
// Turbo encoder
// msg -- {0,1}
// stream -- {0,1}
// puncture -- true to get 1/2 rate,NOT tested yet
//////////////////////////////////////////////////////////////////////
void encode(BYTE *msg, BYTE *stream, bool puncture)
{
	INT i;
	BYTE imsg[L_TOTAL];
	BYTE chkBuffer[2][L_TOTAL];
	// first encoder
	RSC_Encode(msg,chkBuffer[0],L_TOTAL,true);
	// interleave
	for (i=0;i<L_TOTAL;i++)
		imsg[i]=msg[m_Inter_table[i]];
	// second encoder
	RSC_Encode(imsg,chkBuffer[1],L_TOTAL,false);
	// punture
	for (i=0;i<L_TOTAL;i++)
	{
		if(puncture){
			stream[i*2]=msg[i];
			stream[i*2+1]=chkBuffer[i%2][i];
		}else{
			stream[i*3]=msg[i];
			stream[i*3+1]=chkBuffer[0][i];
			stream[i*3+2]=chkBuffer[1][i];
		}	
	}
}


__global__ void interLeave(double * src, double * des , unsigned int * interLeaveTable ){
    const int tid = threadIdx.x;
    des[tid] = src[interLeaveTable[tid]];
}

__global__ void deInterLeave(double * src, double * des , unsigned int * interLeaveTable ){
    const int tid = threadIdx.x;
    des[interLeaveTable[tid]] = src[tid];
}

__global__ void gammaAlpha(double * msg ,double * parity, double * L_a, double (*gamma)[8][8], BYTE (*lastState)[8],char (*lastOut)[8] ){
    const int tid = threadIdx.x;

    unsigned int s0,s2;
    for (s0=0;s0<NSTATE;s0++) {
		for (s2=0;s2<NSTATE;s2++)
			gamma[tid][s0][s2]=-INIFINITY;
		gamma[tid][s0][lastState[0][s0]]=-msg[tid]+parity[tid]*lastOut[0][s0]-log(1+exp(L_a[tid]));
		gamma[tid][s0][lastState[1][s0]]=msg[tid]+parity[tid]*lastOut[1][s0]+L_a[tid]-log(1+exp(L_a[tid]));
		//gamma[tid][s0][lastState[0][s0]]=0.5;
		//gamma[tid][s0][lastState[1][s0]]=-0.5;
    }
}

__global__ void gammaBeta(double * msg ,double * parity, double * L_a, double (*gamma)[8][8], BYTE (*nextState)[8], char (*nextOut)[8]){
    const int tid = threadIdx.x;

    unsigned int s0,s2;
    for (s0=0;s0<NSTATE;s0++) {
		for (s2=0;s2<NSTATE;s2++)
			gamma[tid][s0][s2]=-INIFINITY;
		gamma[tid][s0][nextState[0][s0]]=-msg[tid]+parity[tid]*nextOut[0][s0]-__logf(1+exp(L_a[tid]));
		gamma[tid][s0][nextState[1][s0]]=msg[tid]+parity[tid]*nextOut[1][s0]+L_a[tid]-__logf(1+exp(L_a[tid]));
		//gamma[tid][s0][nextState[0][s0]]=0.5;
		//gamma[tid][s0][nextState[1][s0]]=-0.5;
    }
}

__global__ void Alpha(double (*Alpha)[8], double (*gamma)[8][8]) {
	const int tid = threadIdx.x;

	UINT k, s1, s2;
	double sum;

	if (tid == 0) {
		Alpha[0][0] = 0;
		for (s1=1;s1<NSTATE;s1++)
			Alpha[0][s1]=-INIFINITY;
	}
	else {
		for (s1=0;s1<NSTATE;s1++)
			Alpha[tid*1024][s1]=0;
	}

	for (k=1; k<=L_TOTAL; k++) {
	//for (k=tid*1024+1; k<(tid*1024+1024); k++) {
        for (s2=0;s2<NSTATE;s2++){
            sum = 0.0;
            for (s1=0;s1<NSTATE;s1++) {
                sum+=exp(gamma[k-1][s2][s1]+Alpha[k-1][s1]);
			}
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                Alpha[k][s2]=-INIFINITY;
            else
                Alpha[k][s2]=log(sum);
        }
	}

}

__global__ void Beta(double (*Beta)[8], double (*gamma)[8][8], bool index) {
	const int tid = threadIdx.x;

	UINT k, s1, s2;
	double sum;

	//if (tid == 5) {
		if (index){// true -- terminated,false -- open
        Beta[L_TOTAL][0]=0;
        for (s2=1;s2<NSTATE;s2++)
            Beta[L_TOTAL][s2]=-INIFINITY;
		}
		else 
			for (s2=0;s2<NSTATE;s2++)
				Beta[L_TOTAL][s2]=0;
	//}
	//else {
	//	for (s2=0; s2<NSTATE; s2++)
	//		Beta[(tid+1)*1024][s2]=0;
	//}

    //for (k=(tid+1)*1024;k>(tid*1024);k--) {
    for (k=L_TOTAL;k>0;k--) {

        for (s1=0;s1<NSTATE;s1++) {
            sum = 0.0;
            for (s2=0;s2<NSTATE;s2++) 
                sum += exp(gamma[k][s1][s2] + Beta[k+1][s2]);
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                Beta[k][s1] = -INIFINITY;
            else 
                Beta[k][s1] = log(sum);
        }
	}
}

void computeAlpha(double (*AlphaHost)[8], double (*gamma)[8][8], double *maxBranch) {
    // initialize Alpha & Beta
    AlphaHost[0][0]=0;
	UINT s1,k,s2;
	double sum;
    for (s1=1;s1<NSTATE;s1++)
        AlphaHost[0][s1]=-INIFINITY;

    for (k=1;k<=L_TOTAL;k++){

        for (s2=0;s2<NSTATE;s2++){
            sum = 0;
            for (s1=0;s1<NSTATE;s1++) {
                sum+=exp(gamma[k-1][s2][s1]+AlphaHost[k-1][s1]);
			}
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                AlphaHost[k][s2]=-INIFINITY;
            else
                AlphaHost[k][s2]=log(sum);
        }

		// normalization,prevent overflow
		maxBranch[k]=AlphaHost[k][0];
		for (s2=1;s2<NSTATE;s2++)
			if (AlphaHost[k][s2]>maxBranch[k])
				maxBranch[k]=AlphaHost[k][s2];

		for (s2=0;s2<NSTATE;s2++)
			AlphaHost[k][s2]=AlphaHost[k][s2]-maxBranch[k];
    }
}

void computeBeta(double (*BetaHost)[8], double (*gamma)[8][8], bool index, double * maxBranch){
    // initialize Beta
	UINT s1,k,s2;
	double sum;
    if (index){// true -- terminated,false -- open
        BetaHost[L_TOTAL][0]=0;
        for (s2=1;s2<NSTATE;s2++)
            BetaHost[L_TOTAL][s2]=-INIFINITY;
    }
    else 
        for (s2=0;s2<NSTATE;s2++)
            BetaHost[L_TOTAL][s2]=0;

    for (k=L_TOTAL-1;k>0;k--) {

        for (s1=0;s1<NSTATE;s1++) {
            sum = 0.0;
            for (s2=0;s2<NSTATE;s2++) 
                sum += exp(gamma[k][s1][s2] + BetaHost[k+1][s2]);
            if (sum<MIN)
            //if (sum<=0.000000000000000000000000000001)
                BetaHost[k][s1] = -INIFINITY;
            else 
                BetaHost[k][s1] = log(sum);
        }

		// normalization,prevent overflow
		for (s2=0;s2<NSTATE;s2++)
			BetaHost[k][s2]=BetaHost[k][s2]-maxBranch[k];
    }
}

__global__ void normalizationAlphaAndBeta(double (*Alpha)[8], double (*Beta)[8]) {
    unsigned int tid = threadIdx.x+1; 
    double max_branch;
    max_branch = Alpha[tid][0];
	UINT s2;
    for (s2=1;s2<NSTATE;s2++)
        if (Alpha[tid][s2]>max_branch)
            max_branch = Alpha[tid][s2];

    for (s2=0;s2<NSTATE;s2++) {
        Alpha[tid][s2] = Alpha[tid][s2] - max_branch;

        if (tid != L_TOTAL) 
            Beta[tid][s2] = Beta[tid][s2] - max_branch;
    }

}

__global__ void LLRS(double * msg, double * parity, double * L_a, double (*Alpha)[8], double (*Beta)[8], double * L_all,BYTE (*lastState)[8], char (*lastOut)[8]) {
    unsigned int tid = threadIdx.x; 
    UINT s2;
	double sum0 = 0.0, sum1 = 0.0;
    for (s2=0;s2<NSTATE;s2++) {
        //gamma[LastState[0][s2]]=-msg[tid]+parity[tid]*LastOut[0][s2]-log(1+exp(L_a[tid]));
        //gamma[LastState[1][s2]]=msg[tid]+parity[tid]*LastOut[1][s2]+L_a[tid]-log(1+exp(L_a[tid]));
        //sum0+=exp(gamma[LastState[0][s2]]+Alpha[tid][LastState[0][s2]]+Beta[tid+1][s2]);
        //sum1+=exp(gamma[LastState[1][s2]]+Alpha[tid][LastState[1][s2]]+Beta[tid+1][s2]);
        double gamma0=-msg[tid]+parity[tid]*lastOut[0][s2]-log(1+exp(L_a[tid]));
        double gamma1=msg[tid]+parity[tid]*lastOut[1][s2]+L_a[tid]-log(1+exp(L_a[tid]));
        sum0+=exp(gamma0+Alpha[tid][lastState[0][s2]]+Beta[tid+1][s2]);
        sum1+=exp(gamma1+Alpha[tid][lastState[1][s2]]+Beta[tid+1][s2]);
    }
    //L_all[tid]=log(sum1)-log(sum0);
    L_all[tid]=log(sum1)-log(sum0);
}

__global__ void extrinsicInformation(double * L_all, double * msg, double * L_a, double * L_e) {
    unsigned int tid = threadIdx.x;
    L_e[tid] = L_all[tid] - 2*msg[tid] - L_a[tid];
}

__global__ void demultiplex(double * stream, double * msg, double * parity0, double * parity1) {
    unsigned int tid = threadIdx.x;
    //if (puncture){// punctured rate=1/2
    //    msg[tid]=stream[2*tid];
    //    parity[tid%2][tid]=stream[tid*2+1];
    //}
    //else {// unpunctured rate=1/3
    //    msg[tid]=stream[3*tid];
    //    parity0[tid]=stream[3*tid+1];
    //    parity1[tid]=stream[3*tid+2];
    //}
        msg[tid]=stream[3*tid];
        parity0[tid]=stream[3*tid+1];
        parity1[tid]=stream[3*tid+2];
}

__global__ void initializeExtrinsicInformation(double * L_e) {
    unsigned int tid = threadIdx.x;
    L_e[tid] = 0;
    
}

__global__ void exestimateInformationBits(double * L_all, BYTE * msghat, UINT * m_Inter_table) {
    unsigned int tid = threadIdx.x;
    if(L_all[tid]>0)
        msghat[m_Inter_table[tid]]=1;
    else
        msghat[m_Inter_table[tid]]=0;
}

void countErrors(BYTE *m, BYTE * mhat, UINT * bitsError, UINT * frameError, UINT iter) {

	bool f_err = false;
	for (int i=0; i<(L_TOTAL-M);i++) {
		if (m[i] != mhat[i]) {
			bitsError[iter] = bitsError[iter]+1;
			f_err = true;
		}
	}

	if (f_err) 
		frameError[iter] = frameError[iter]+1;
}


int main(int argc, char* argv[])
{
    initRandom(seed);

	BYTE * m;
	BYTE * x;
	double * y;
	BYTE * mhat;

	int frame;
	UINT bits_all,bits_err[MAXITER],frame_err[MAXITER];
	double Ber,Fer;
	double Eb_No_dB,No;
	//bool f_err;
	//FILE * fp;
	int i;

	m = new BYTE[L_TOTAL];
	x = new BYTE[L_ALL];
	y = new double[L_ALL];
	mhat = new BYTE[L_TOTAL];

	init_Block_interleave_table();	// block interleave


    
    findCudaDevice(argc, (const char **)argv);

	BYTE (*LastStateDevice)[8];
	BYTE (*NextStateDevice)[8];
	char (*LastOutDevice)[8];
	char (*NextOutDevice)[8];
	double * yDevice;
	double * msgDevice;
	double * imsgDevice;
	BYTE * mhatDevice;
	double * parity0Device;
	double * parity1Device;
	UINT * tableDevice;
	double * L_eDevice;
	double * L_aDevice;
	double * L_allDevice;
	double (*gammaAlphaDevice)[8][8];
	double (*gammaBetaDevice)[8][8];
	double (*AlphaDevice)[8];
	double (*BetaDevice)[8];

    hipMalloc((void **)&LastStateDevice, 2*8*sizeof(BYTE));
    hipMalloc((void **)&NextStateDevice, 2*8*sizeof(BYTE));
    hipMalloc((void **)&LastOutDevice, 2*8*sizeof(char));
    hipMalloc((void **)&NextOutDevice, 2*8*sizeof(char));

    hipMalloc((void **)&yDevice, L_ALL*sizeof(double));
    hipMalloc((void **)&msgDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&imsgDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&mhatDevice, L_TOTAL*sizeof(BYTE));
    hipMalloc((void **)&parity0Device, L_TOTAL*sizeof(double));
    hipMalloc((void **)&parity1Device, L_TOTAL*sizeof(double));
    hipMalloc((void **)&tableDevice, L_TOTAL*sizeof(unsigned int));
    hipMalloc((void **)&L_eDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&L_aDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&L_allDevice, L_TOTAL*sizeof(double));
    hipMalloc((void **)&gammaAlphaDevice, L_TOTAL*sizeof(double)*8*8);
    hipMalloc((void **)&gammaBetaDevice, L_TOTAL*sizeof(double)*8*8);
    hipMalloc((void **)&AlphaDevice, (L_TOTAL+1)*sizeof(double)*8);
    hipMalloc((void **)&BetaDevice, (L_TOTAL+1)*sizeof(double)*8);

	//For Debug
	//double L_aHost[L_TOTAL];
	//double L_aHost1[L_TOTAL];
	//double L_allHost[L_TOTAL];

    double gammaAlphaHost[L_TOTAL][8][8];
    double gammaBetaHost[L_TOTAL][8][8];
    double AlphaHost[L_TOTAL+1][8];
    double BetaHost[L_TOTAL+1][8];

	double max_branch[L_TOTAL+1];

    hipMemcpy(LastStateDevice,LastState,sizeof(BYTE)*2*8, hipMemcpyHostToDevice);
    hipMemcpy(NextStateDevice,NextState,sizeof(BYTE)*2*8, hipMemcpyHostToDevice);
    hipMemcpy(LastOutDevice,LastOut,sizeof(char)*2*8, hipMemcpyHostToDevice);
    hipMemcpy(NextOutDevice,NextOut,sizeof(char)*2*8, hipMemcpyHostToDevice);

    hipMemcpy(tableDevice,m_Inter_table,sizeof(unsigned int)*L_TOTAL, hipMemcpyHostToDevice);

	for (Eb_No_dB= 0.0;Eb_No_dB<5.0;Eb_No_dB+=0.5){

	//Eb_No_dB = 0.0;
		No = 1/pow(10.0,Eb_No_dB/10.0);
		bits_all = 0;
		for (i =0; i<MAXITER;i++) {
			bits_err[i]=0;
			frame_err[i]=0;
		}

		for (frame = 0; frame<FRAME_NUM; frame++, bits_all += (L_TOTAL-M)) {

			// Generate random information bits
			for (i=0;i<L_TOTAL;i++)
				if (boolrandom())
					m[i]=1;
				else
					m[i]=0;
			// encoder
			encode(m,x,false);
			// add noise
			for (i=0;i<L_ALL;i++)
				if (x[i])
					y[i]=1.0+gaussian(No/2);
				else
					y[i]=-1.0+gaussian(No/2);

			hipMemcpy(yDevice,y,sizeof(double)*L_ALL, hipMemcpyHostToDevice);

			demultiplex<<<1,L_TOTAL>>>(yDevice, msgDevice, parity0Device, parity1Device); 
			interLeave<<<1,L_TOTAL>>>(msgDevice, imsgDevice, tableDevice);
			initializeExtrinsicInformation<<<1,L_TOTAL>>>(L_eDevice);

			for (int iter = 0; iter<MAXITER; iter++) {
				
				deInterLeave<<<1,L_TOTAL>>>(L_eDevice, L_aDevice, tableDevice);

				gammaAlpha<<<1,L_TOTAL>>>(msgDevice , parity0Device,  L_aDevice,  gammaAlphaDevice,LastStateDevice, LastOutDevice);
				gammaBeta<<<1,L_TOTAL>>>(msgDevice , parity0Device,  L_aDevice,  gammaBetaDevice, NextStateDevice, NextOutDevice);
				//Alpha<<<1,1>>>(AlphaDevice, gammaAlphaDevice);
				//Beta<<<1,1>>>(BetaDevice, gammaBetaDevice,true);
				//hipMemcpy(AlphaHost, AlphaDevice, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyDeviceToHost);
				hipMemcpy(gammaAlphaHost, gammaAlphaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);
				hipMemcpy(gammaBetaHost, gammaBetaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);

				computeAlpha(AlphaHost, gammaAlphaHost, max_branch);
				computeBeta(BetaHost, gammaBetaHost, true,max_branch);
				hipMemcpy(AlphaDevice, AlphaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				hipMemcpy(BetaDevice, BetaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				//normalizationAlphaAndBeta<<<1,L_TOTAL>>>(AlphaDevice, BetaDevice);

				LLRS<<<1,L_TOTAL>>>(msgDevice, parity0Device, L_aDevice, AlphaDevice, BetaDevice, L_allDevice,LastStateDevice, LastOutDevice);

				extrinsicInformation<<<1, L_TOTAL>>>(L_allDevice, msgDevice, L_aDevice, L_eDevice);
				//if (iter >= 3) {
				///debug
				//exestimateInformationBits<<<1,L_TOTAL>>>(L_allDevice, mhatDevice, tableDevice); 

				//hipMemcpy(mhat, mhatDevice, sizeof(BYTE)*L_TOTAL, hipMemcpyDeviceToHost);
				//countErrors(m, mhat, bits_err, frame_err, iter);
				//debug
				//hipMemcpy(L_aHost1, L_aDevice, sizeof(double)*L_TOTAL, hipMemcpyDeviceToHost);
				//}

				interLeave<<<1, L_TOTAL>>>(L_eDevice, L_aDevice, tableDevice);

				gammaAlpha<<<1,L_TOTAL>>>(imsgDevice , parity1Device,  L_aDevice,  gammaAlphaDevice, LastStateDevice, LastOutDevice);
				gammaBeta<<<1,L_TOTAL>>>(imsgDevice , parity1Device,  L_aDevice,  gammaBetaDevice, NextStateDevice, NextOutDevice);
				Alpha<<<1,1>>>(AlphaDevice, gammaAlphaDevice);
				Beta<<<1,1>>>(BetaDevice, gammaBetaDevice,false);
				//hipMemcpy(gammaAlphaHost, gammaAlphaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);
				//hipMemcpy(gammaBetaHost, gammaBetaDevice, sizeof(double)*L_TOTAL*8*8, hipMemcpyDeviceToHost);

				//computeAlpha(AlphaHost, gammaAlphaHost, max_branch);
				//Beta(BetaHost, gammaBetaHost, false, max_branch);
				//hipMemcpy(AlphaDevice, AlphaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				//hipMemcpy(BetaDevice, BetaHost, sizeof(double)*(L_TOTAL+1)*8, hipMemcpyHostToDevice);
				normalizationAlphaAndBeta<<<1,L_TOTAL>>>(AlphaDevice, BetaDevice);

				LLRS<<<1,L_TOTAL>>>(imsgDevice, parity1Device, L_aDevice, AlphaDevice, BetaDevice, L_allDevice, LastStateDevice,LastOutDevice);

				extrinsicInformation<<<1, L_TOTAL>>>(L_allDevice, imsgDevice, L_aDevice, L_eDevice);

				exestimateInformationBits<<<1,L_TOTAL>>>(L_allDevice, mhatDevice, tableDevice); 

				hipMemcpy(mhat, mhatDevice, sizeof(BYTE)*L_TOTAL, hipMemcpyDeviceToHost);
				countErrors(m, mhat, bits_err, frame_err, iter);

				//debug
				//hipMemcpy(L_aHost, L_aDevice, sizeof(double)*L_TOTAL, hipMemcpyDeviceToHost);
			}
			// estimate information bits
			//exestimateInformationBits<<<1,L_TOTAL>>>(L_allDevice, mhatDevice, tableDevice); 

			//hipMemcpy(mhat, mhatDevice, sizeof(BYTE)*L_TOTAL, hipMemcpyDeviceToHost);
			// count errors
			//UINT bits_err = 0;
			//for (i=0;i<L_TOTAL-M;i++) {
			//	if (mhat[i]!=m[i]) {
			//		bits_err++;
			//	}
			//}
			//cout<<"bits_err: "<<bits_err<<endl;
		}

		printf("-------------------------\n");
		printf("Eb/No=%fdB:\n",Eb_No_dB);
		printf("-------------------------\n");
		//fprintf(fp,"-------------------------\n");
		//fprintf(fp,"Eb/No=%fdB:\n",Eb_No_dB);
		//fprintf(fp,"-------------------------\n");

		for (i=0;i<MAXITER;i++) {
			Ber=(double)bits_err[i]/(double)bits_all;
			Fer=(double)frame_err[i]/(double)FRAME_NUM;
			printf("Iteration:%d\n",i+1);
			printf("---Ber=%f\n---Fer=%f\n",Ber,Fer);
			//fprintf(fp,"Iteration:%d\n",i);
			//fprintf(fp,"---Ber=%f\n---Fer=%f\n",Ber,Fer);
		}
	}

	delete m;
	delete x;
	delete y;
	delete mhat;

	hipFree(LastStateDevice);
	hipFree(NextStateDevice);
	hipFree(LastOutDevice);
	hipFree(NextOutDevice);
	hipFree(yDevice);
	hipFree(msgDevice);
	hipFree(imsgDevice);
	hipFree(mhatDevice);
	hipFree(parity0Device);
	hipFree(parity1Device);
	hipFree(tableDevice);
	hipFree(L_eDevice);
	hipFree(L_aDevice);
	hipFree(L_allDevice);
	hipFree(gammaAlphaDevice);
	hipFree(gammaBetaDevice);
	hipFree(AlphaDevice);
	hipFree(BetaDevice);

	return 0;
}
